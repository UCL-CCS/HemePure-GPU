#include "hip/hip_runtime.h"

// This file is part of the GPU development for HemeLB
// 7-1-2019
/**
	Contains the GPU cuda kernels for the Iolet type of collision-streaming,
	i.e. for the Inlet and outlet collision-streaming
	and the 2 types of iolet BCs:
	 	1. Velocity BCs (LADDIOLET option in CMake file)
		2. Pressure BCs (NASHZEROTHORDERPRESSUREIOLET option in CMake file)
*/

#include <stdio.h>
#include "units.h"

#ifdef HEMELB_USE_GPU
#include "cuda_kernels_def_decl/cuda_params.h"
//#include <thrust/host_vector.h>
//#include <thrust/device_vector.h>
//#include <thrust/copy.h>
//#include <thrust/fill.h>
#endif


namespace hemelb
{

#ifdef HEMELB_USE_GPU



	//**************************************************************
	// Kernel for the Collision step for the Lattice Boltzmann algorithm
	// Collision Type 3: mInletCollision: Inlet BCs
	//
	// Inlet BCs: specified with HEMELB_INLET_BOUNDARY in CMakeLists.txt
 	//	Two Possible types of Inlet BCs:
	// 	1. NashZerothOrderPressure: Implement this first (see lb/streamers/NashZerothOrderPressureDelegate.h)
	//	2. LaddIolet: (see lb/streamers/LaddIoletDelegate.h)
	//
	// Implementation currently follows the memory arrangement of the data
	// by index LB, i.e. method (b)
	// Need to pass the information for the fluid-iolet links - To do!!!
	// This information is in ioletIntersection, see geometry/SiteDataBare.h
	//
	//**************************************************************
	__global__ void GPU_CollideStream_3_NashZerothOrderPressure(distribn_t* GMem_dbl_fOld_b,
																distribn_t* GMem_dbl_fNew_b,
																distribn_t* GMem_dbl_MacroVars,
																int64_t* GMem_int64_Neigh,
																uint32_t* GMem_uint32_Iolet_Link,
																distribn_t* GMem_ghostDensity,
																float* GMem_inletNormal,
																int nInlets,
																uint64_t nArr_dbl,
																uint64_t lower_limit, uint64_t upper_limit, uint64_t totalSharedFs)
	{
		unsigned long long Ind = blockIdx.x * blockDim.x + threadIdx.x;
		Ind = Ind + lower_limit;

		if(Ind >= upper_limit)
			return;

		// Load the distribution functions
		//f[19] and fEq[19]
		double dev_ff[19], dev_fEq[19];
		double nn = 0.0;	// density
		double momentum_x, momentum_y, momentum_z;
		momentum_x = momentum_y = momentum_z = 0.0;

		double velx, vely, velz;	// Fluid Velocity

		for(int i=0; i< _NUMVECTORS; i++){
			dev_ff[i] = GMem_dbl_fOld_b[(unsigned long long)i * nArr_dbl + Ind];
		}
	//	__syncthreads(); // Check if this is needed or maybe I can have the density calculation within the loop


		//-----------------------------------------------------------------------------------------------------------
		// Calculate the nessessary elements for calculating the equilibrium distribution functions
		// a. Calculate density
		// b. Calculate momentum - Needs to consider the case of body force as well - To do!!!
		for(int direction = 0; direction< _NUMVECTORS; direction++){
			nn += dev_ff[direction];
			momentum_x += (double)_CX_19[direction] * dev_ff[direction];
			momentum_y += (double)_CY_19[direction] * dev_ff[direction];
			momentum_z += (double)_CZ_19[direction] * dev_ff[direction];
			//printf("Momentum: _x = %.5e, _y = %.5e, _z = %.5e \n\n", momentum_x, momentum_y, momentum_z);
		}


		// In the case of body force
		//momentum_x += 0.5 * _force_x;
		//momentum_y += 0.5 * _force_y;
		//momentum_z += 0.5 * _force_z;


		// Compute velocity components
		velx = momentum_x/nn;
		vely = momentum_y/nn;
		velz = momentum_z/nn;

		//-----------------------------------------------------------------------------------------------------------
		// c. Calculate equilibrium distr. functions
		double density_1 = 1.0 / nn;
		double momentumMagnitudeSquared = momentum_x * momentum_x
													+ momentum_y * momentum_y + momentum_z * momentum_z;

		for (int i = 0; i < _NUMVECTORS; ++i)
		{
			double mom_dot_ei = (double)_CX_19[i] * momentum_x
									+ (double)_CY_19[i] * momentum_y
									+ (double)_CZ_19[i] * momentum_z;

			dev_fEq[i] = _EQMWEIGHTS_19[i]
							* (nn - (3.0 / 2.0) * momentumMagnitudeSquared * density_1
											+ (9.0 / 2.0) * density_1 * mom_dot_ei * mom_dot_ei + 3.0 * mom_dot_ei);
		}
		//-----------------------------------------------------------------------------------------------------------

		// d. Body Force case: Add details of any forcing scheme here - Evaluate force[i]
		// To do!!!
		//-----------------------------------------------------------------------------------------------------------

		// Collision step:
		// Single Relaxation Time approximation (LBGK)
		double dev_fn[19];		// or maybe use the existing dev_ff[_NUMVECTORS] to minimise the memory requirements - Check and replace in the future

		// Evolution equation for the fi's here
		for (int i = 0; i < _NUMVECTORS; ++i)
		{
			dev_fn[i] = dev_ff[i] + (dev_fEq[i] - dev_ff[i])/dev_tau; // + force[i];
		}

		//__syncthreads(); // Check if needed!


		// --------------------------------------------------------------------------------
		// Streaming Step:
		// a. Load the streaming indices
		// b. The Iolet-Fluid links info
		// c. The ghost density
		// d. The inletNormal


		// a. Bulk Streaming indices: dev_NeighInd[19] here refers to either: a) the ACTUAL fluid ID index or b) the hemeLB neighbourIndices which refer to the array Index (Data Address) in f_old and f_new
		int64_t dev_NeighInd[19]; // ACTUAL fluid ID index for the neighbours - or streaming Data Address in hemeLB f's memory

		// printf("Number of inlets: %d \n\n", nInlets);
		distribn_t ghost_dens; // = 0.0; //new distribn_t[nInlets];	// c. The ghost density
		float inletNormal_x, inletNormal_y, inletNormal_z;


		for(int LB_Dir=0; LB_Dir< _NUMVECTORS; LB_Dir++){

			// If we use the elements in GMem_int64_Neigh - then we access the memory address in fOld or fNew directly (not the fluid id)
			// (remember the memory layout in hemeLB is based on the site fluid index, i.e. f0[0], f1[0], f2[0], ..., fq[0] and for the Fluid Index Ind : f0[Ind], f1[Ind], f2[Ind], ..., fq[Ind]

			dev_NeighInd[LB_Dir] = GMem_int64_Neigh[(unsigned long long)LB_Dir * nArr_dbl + Ind]; // Read the streaming info here - Here Refers to Data Address NOT THE STREAMING FLUID ID!!!

		}

		//__syncthreads();

		//
		// b. Iolet-Fluid links info:
		uint32_t Iolet_Intersect = GMem_uint32_Iolet_Link[Ind];

		/*
		ghost_dens[0] = GMem_ghostDensity[0];

		inletNormal[0] = GMem_inletNormal[0];
		inletNormal[1] = GMem_inletNormal[1];
		inletNormal[2] = GMem_inletNormal[2];
		*/


		// Read the ghost density and the inlet Normal
		// How do I distinguish which inlet ID do I have ??? Need to think about this... To do!!!
		// Need to pass this info based on the site Index (from the initialisation process. With given site ranges -> int boundaryId = site.GetIoletId();)

		// Find a way to pass the IdInlet - To do!!!
		// Place that here:
		int IdInlet=0; // This will be replaced by whatever way we manage to read the inlet ID based on maybe fluid ID

		ghost_dens = GMem_ghostDensity[IdInlet];
		inletNormal_x = GMem_inletNormal[3*IdInlet];
		inletNormal_y = GMem_inletNormal[3*IdInlet+1];
		inletNormal_z = GMem_inletNormal[3*IdInlet+2];

	/*
		for (int IdInlet=0; IdInlet<nInlets; IdInlet++) {
			ghost_dens[IdInlet] = GMem_ghostDensity[IdInlet];
			//printf("IdInlet: %d, ghost_dens[%d]: %.5f \n\n", IdInlet, IdInlet, ghost_dens[IdInlet]);

			inletNormal[3*IdInlet] = GMem_inletNormal[3*IdInlet];
			inletNormal[3*IdInlet+1] = GMem_inletNormal[3*IdInlet+1];
			inletNormal[3*IdInlet+2] = GMem_inletNormal[3*IdInlet+2];
		}
	*/
		// printf("ghost_dens[0]: %.5f, inletNormal_x = %.5f, inletNormal_y = %.5f, inletNormal_z = %.5f  \n\n", ghost_dens[0], inletNormal[0], inletNormal[1], inletNormal[2]);

		//__syncthreads();


		// Put the new populations after collision in the GMem_dbl array,
		// implementing the streaming step with Simple Bounce Back if Wall-Fluid link

		// fNew (dev_fn) populations:
		for (int LB_Dir = 0; LB_Dir < _NUMVECTORS; LB_Dir++)
		{
			unsigned mask = 1U << (LB_Dir - 1); // Needs to left shift the bits in mask so that I can then compare against the value in test_Wall_Intersect (To do: compare against test_bool_Wall_Intersect as well)
			bool is_Iolet_link = (Iolet_Intersect & mask);

			if(is_Iolet_link){	// ioletLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);

				//=============================================================================================================
				// Not valid in general! Need to change!!!
				// (IdInlet=0) Here we assume that we have only one inlet and the value of int boundaryId = site.GetIoletId() = 1. Need to change in the future!!!
				double component = velx*inletNormal_x + vely*inletNormal_y + velz*inletNormal_z;	// distribn_t component = (hydroVars.momentum / hydroVars.density).Dot(ioletNormal);

				// ghostHydrovars.momentum = ioletNormal * component * ghostDensity;
				double momentum_x = inletNormal_x * component * ghost_dens;
				double momentum_y = inletNormal_y * component * ghost_dens;
				double momentum_z = inletNormal_z * component * ghost_dens;


				//------------------------------------------------------------------------------------------------------
				// Calculate Feq[unstreamed_dir] - Only the direction that is necessary
				density_1 = 1.0 / ghost_dens;
				momentumMagnitudeSquared = momentum_x * momentum_x
													+ momentum_y * momentum_y + momentum_z * momentum_z;

				int unstreamed_dir = _InvDirections_19[LB_Dir];
				double mom_dot_ei = (double)_CX_19[unstreamed_dir] * momentum_x
									+ (double)_CY_19[unstreamed_dir] * momentum_y
									+ (double)_CZ_19[unstreamed_dir] * momentum_z;

				double dev_fEq_unstr = _EQMWEIGHTS_19[unstreamed_dir]
							* (ghost_dens - (3.0 / 2.0) * momentumMagnitudeSquared * density_1
											+ (9.0 / 2.0) * density_1 * mom_dot_ei * mom_dot_ei + 3.0 * mom_dot_ei);
				//------------------------------------------------------------------------------------------------------
				// Need to distinguish the int boundaryId = site.GetIoletId() correctly and pass the info (identify the proper ghost density and inlet-normals.
				//=============================================================================================================

				// printf("Site ID = %lld - Inlet in Dir: %d, Unstreamed direction: %d, fEq = %.5e \n\n", Ind, LB_Dir, unstreamed_dir, dev_fEq_unstr);

				// Case of NashZerothOrderPressure:
				// *latticeData->GetFNew(site.GetIndex() * LatticeType::NUMVECTORS + unstreamed) = ghostHydrovars.GetFEq()[unstreamed];
				GMem_dbl_fNew_b[(unsigned long long)unstreamed_dir * nArr_dbl + Ind] = dev_fEq_unstr;

			}
			else{ // bulkLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);
				//dev_NeighInd[LB_Dir] = GMem_int64_Neigh[(unsigned long long)LB_Dir * nArr_dbl + Ind]; // Read the streaming info here - Here Refers to Data Address NOT THE STREAMING FLUID ID!!!
				//---------------------------------------------------------------------------
				// If it streams in direction inside the simulation domain then it will point to a fluid ID < nFluid_nodes, otherwise it will stream to a neighbouring rank (place in the totalSharedFs at the end of the array)
				if (dev_NeighInd[LB_Dir] < (nArr_dbl*_NUMVECTORS) ) // maximum Data Address in array that correspond to this domain = nFluid_nodes*_NUMVECTORS
				{
					dev_NeighInd[LB_Dir] = (dev_NeighInd[LB_Dir] - LB_Dir)/_NUMVECTORS;	// Evaluate the ACTUAL streaming fluid ID index

					// Save the post collision population in fNew
					GMem_dbl_fNew_b[(unsigned long long)LB_Dir * nArr_dbl + dev_NeighInd[LB_Dir]] = dev_fn[LB_Dir];
				}
				else{
					// Save the post collision population in fNew[Addr] at the end of the array in the (1+totalSharedFs) location
					GMem_dbl_fNew_b[dev_NeighInd[LB_Dir]] = dev_fn[LB_Dir];

					//
					// Debugging - Remove later
					// Check if it points to an address outside the (nFluid_nodes * _NUMVECTORS + 1+totalSharedFs )
					if (dev_NeighInd[LB_Dir] >= (nArr_dbl*_NUMVECTORS+1+totalSharedFs)) printf("Error!!! Fluid Index = %lld, Stream.Dir.= %d, Max. Streaming addr = %lld Vs Stream. Addr.=%lld \n\n", Ind, LB_Dir, nArr_dbl*_NUMVECTORS+1+totalSharedFs, dev_NeighInd[LB_Dir] );
					//
				}

				//---------------------------------------------------------------------------
			}

		}

		//=============================================================================================

		// Write old density and velocity to memory -
		// Maybe use a different cuda kernel for these calculations (if saving the MacroVariables delays the collision/streaming kernel)
		// Check -  To do!!!
		GMem_dbl_MacroVars[Ind] = nn;
		GMem_dbl_MacroVars[1ULL*nArr_dbl + Ind] = velx;
		GMem_dbl_MacroVars[2ULL*nArr_dbl + Ind] = vely;
		GMem_dbl_MacroVars[3ULL*nArr_dbl + Ind] = velz;

	} // Ends the kernel GPU_Collide Type 2: mWallCollision: Case Fluid-Wall collision
	//==========================================================================================


	//**************************************************************
	// Kernel for the Collision step for the Lattice Boltzmann algorithm
	// 	Collision Type 3: mInletCollision: Inlet BCs
	//	Collision Type 4: mOutletCollision: Outlet BCs
	//
	// Inlet BCs: specified with HEMELB_INLET_BOUNDARY in CMakeLists.txt
 	//	Two Possible types of Inlet BCs:
	// 	1. NashZerothOrderPressure: Implement this first (see lb/streamers/NashZerothOrderPressureDelegate.h)
	//	2. LaddIolet: (see lb/streamers/LaddIoletDelegate.h)
	//
	// Implementation currently follows the memory arrangement of the data
	// by index LB, i.e. method (b)
	// Need to pass the information for the fluid-iolet links - Done!!!
	// This information is in ioletIntersection, see geometry/SiteDataBare.h
	//
	// This version uses the ACTUAL streaming address in global memory - NOT the fluid ID.
	//**************************************************************
	__global__ void GPU_CollideStream_3_NashZerothOrderPressure_new(distribn_t* GMem_dbl_fOld_b,
																distribn_t* GMem_dbl_fNew_b,
																distribn_t* GMem_dbl_MacroVars,
																int64_t* GMem_int64_Neigh,
																uint32_t* GMem_uint32_Iolet_Link,
																distribn_t* GMem_ghostDensity,
																float* GMem_inletNormal,
																int nInlets,
																uint64_t nArr_dbl,
																uint64_t lower_limit, uint64_t upper_limit, uint64_t totalSharedFs, int time_Step, int num_local_Iolets, site_t* iolets_ID_range)
	{
		unsigned long long Ind = blockIdx.x * blockDim.x + threadIdx.x;
		Ind = Ind + lower_limit;

		if(Ind >= upper_limit)
			return;

		// Load the distribution functions
		//f[19] and fEq[19]
		double dev_ff[19], dev_fEq[19];
		double nn = 0.0;	// density
		double momentum_x, momentum_y, momentum_z;
		momentum_x = momentum_y = momentum_z = 0.0;

		double velx, vely, velz;	// Fluid Velocity

		//-----------------------------------------------------------------------------------------------------------
		// 1. Read the fOld_GPU_b distr. functions
		// 2. Calculate the nessessary elements for calculating the equilibrium distribution functions
		// 		a. Calculate density
		// 		b. Calculate momentum - Needs to consider the case of body force as well - To do!!!
		for(int direction = 0; direction< _NUMVECTORS; direction++){
			dev_ff[direction] = GMem_dbl_fOld_b[(unsigned long long)direction * nArr_dbl + Ind];

			nn += dev_ff[direction];
			momentum_x += (double)_CX_19[direction] * dev_ff[direction];
			momentum_y += (double)_CY_19[direction] * dev_ff[direction];
			momentum_z += (double)_CZ_19[direction] * dev_ff[direction];
			//printf("Momentum: _x = %.5e, _y = %.5e, _z = %.5e \n\n", momentum_x, momentum_y, momentum_z);
		}


		// In the case of body force
		//momentum_x += 0.5 * _force_x;
		//momentum_y += 0.5 * _force_y;
		//momentum_z += 0.5 * _force_z;


		// Compute velocity components
		velx = momentum_x/nn;
		vely = momentum_y/nn;
		velz = momentum_z/nn;

		//-----------------------------------------------------------------------------------------------------------
		// c. Calculate equilibrium distr. functions
		double density_1 = 1.0 / nn;
		double momentumMagnitudeSquared = momentum_x * momentum_x
													+ momentum_y * momentum_y + momentum_z * momentum_z;

		for (int i = 0; i < _NUMVECTORS; ++i)
		{
			double mom_dot_ei = (double)_CX_19[i] * momentum_x
									+ (double)_CY_19[i] * momentum_y
									+ (double)_CZ_19[i] * momentum_z;

			dev_fEq[i] = _EQMWEIGHTS_19[i]
							* (nn - (3.0 / 2.0) * momentumMagnitudeSquared * density_1
											+ (9.0 / 2.0) * density_1 * mom_dot_ei * mom_dot_ei + 3.0 * mom_dot_ei);
		}
		//-----------------------------------------------------------------------------------------------------------

		// d. Body Force case: Add details of any forcing scheme here - Evaluate force[i]
		// To do!!!
		//-----------------------------------------------------------------------------------------------------------

		// Collision step:
		// Single Relaxation Time approximation (LBGK)
		//double dev_fn[19];		// or maybe use the existing dev_ff[_NUMVECTORS] to minimise the memory requirements - Check and replace in the future

		// Evolution equation for the fi's here
		for (int i = 0; i < _NUMVECTORS; ++i)
		{
			//dev_fn[i] = dev_ff[i] + (dev_fEq[i] - dev_ff[i])/dev_tau; // + force[i];
			dev_ff[i] += (dev_ff[i] - dev_fEq[i]) * dev_minusInvTau; // Check if multiplying by dev_minusInvTau makes a difference
		}
		//__syncthreads(); // Check if needed!


		// --------------------------------------------------------------------------------
		// Streaming Step:
		// a. Load the streaming indices
		// b. The Iolet-Fluid links info
		// c. The ghost density
		// d. The inletNormal

		// a. Bulk Streaming indices: dev_NeighInd[19] here refers to the ACTUAL Streaming Array index (Data Address) in f_old and f_new
		//int64_t dev_NeighInd[19];

		// printf("Number of inlets: %d \n\n", nInlets);
		distribn_t ghost_dens; // = 0.0; //new distribn_t[nInlets];	// c. The ghost density
		float inletNormal_x, inletNormal_y, inletNormal_z;

/*
		for(int LB_Dir=0; LB_Dir< _NUMVECTORS; LB_Dir++){

			// If we use the elements in GMem_int64_Neigh - then we access the memory address in fOld or fNew directly (not the fluid id)
			// (remember the memory layout in hemeLB is based on the site fluid index, i.e. f0[0], f1[0], f2[0], ..., fq[0] and for the Fluid Index Ind : f0[Ind], f1[Ind], f2[Ind], ..., fq[Ind]

			dev_NeighInd[LB_Dir] = GMem_int64_Neigh[(unsigned long long)LB_Dir * nArr_dbl + Ind]; // Read the streaming info here - Here Refers to Data Address NOT THE STREAMING FLUID ID!!!
		}
		__syncthreads();
*/
		//
		// b. Iolet-Fluid links info:
		uint32_t Iolet_Intersect = GMem_uint32_Iolet_Link[Ind];


		// Read the ghost density and the inlet Normal
		// How do I distinguish which inlet ID do I have ??? Need to think about this... To do!!!
		// Need to pass this info based on the site Index (from the initialisation process. With given site ranges -> int boundaryId = site.GetIoletId();)

		// Determine the IdInlet - Done!!!
		int IdInlet = INT32_MAX; // Iolet (Inlet/Outlet) ID
		if(num_local_Iolets==1){
			IdInlet = iolets_ID_range[0];
		}
		else{
			// Call a device function to determine which is the Iolet ID - using the iolets_ID_range Array
			// iolets_ID_range Array:
			//	a. Size: num_local_Iolets * 3
			// 	b. Iolet ID, Range of fluid IDs: [lower_limit, upper_limit)
			_determine_Iolet_ID(num_local_Iolets, iolets_ID_range, Ind, &IdInlet);
		}

		// Testing:
		if(IdInlet==INT32_MAX)
		{
			printf("Fluid_ID : %lld, ID_iolet: %d - Fluid NOT in IOLET range!!! \n\n", Ind, IdInlet);
		}
/*		else{
			printf("Fluid_ID : %lld, ID_iolet: %d \n\n", Ind, IdInlet);
		}*/

		//printf("Number of local Iolets = %d \n", num_local_Iolets);
/*
		for (int index = 0; index < num_local_Iolets; index++){
			printf(" Iolet ID: %d, lower_range: %lld, upper_range: %lld ", iolets_ID_range[3*index], iolets_ID_range[3*index+1], iolets_ID_range[3*index+2]);
		}
		printf("\n\n");
*/

		ghost_dens = GMem_ghostDensity[IdInlet];
		inletNormal_x = GMem_inletNormal[3*IdInlet];
		inletNormal_y = GMem_inletNormal[3*IdInlet+1];
		inletNormal_z = GMem_inletNormal[3*IdInlet+2];
//		printf("ghost_dens[%d]: %.5f, inletNormal_x = %.5f, inletNormal_y = %.5f, inletNormal_z = %.5f  \n\n", IdInlet, ghost_dens, inletNormal_x, inletNormal_y, inletNormal_z);


		// Put the new populations after collision in the GMem_dbl array,
		// implementing the streaming step with Simple Bounce Back if Wall-Fluid link

		// fNew (dev_fn) populations:
		for (int LB_Dir = 0; LB_Dir < _NUMVECTORS; LB_Dir++)
		{
			unsigned mask = 1U << (LB_Dir - 1); // Needs to left shift the bits in mask so that I can then compare against the value in test_Wall_Intersect (To do: compare against test_bool_Wall_Intersect as well)
			bool is_Iolet_link = (Iolet_Intersect & mask);

			if(is_Iolet_link){	// ioletLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);

				//=============================================================================================================
				// Not valid in general! Need to change!!!
				// (IdInlet=0) Here we assume that we have only one inlet and the value of int boundaryId = site.GetIoletId() = 1. Need to change in the future!!!
				double component = velx*inletNormal_x + vely*inletNormal_y + velz*inletNormal_z;	// distribn_t component = (hydroVars.momentum / hydroVars.density).Dot(ioletNormal);

				// ghostHydrovars.momentum = ioletNormal * component * ghostDensity;
				double momentum_x = inletNormal_x * component * ghost_dens;
				double momentum_y = inletNormal_y * component * ghost_dens;
				double momentum_z = inletNormal_z * component * ghost_dens;


				//------------------------------------------------------------------------------------------------------
				// Calculate Feq[unstreamed_dir] - Only the direction that is necessary
				density_1 = 1.0 / ghost_dens;
				momentumMagnitudeSquared = momentum_x * momentum_x
													+ momentum_y * momentum_y + momentum_z * momentum_z;

				int unstreamed_dir = _InvDirections_19[LB_Dir];
				double mom_dot_ei = (double)_CX_19[unstreamed_dir] * momentum_x
									+ (double)_CY_19[unstreamed_dir] * momentum_y
									+ (double)_CZ_19[unstreamed_dir] * momentum_z;

				double dev_fEq_unstr = _EQMWEIGHTS_19[unstreamed_dir]
							* (ghost_dens - (3.0 / 2.0) * momentumMagnitudeSquared * density_1
											+ (9.0 / 2.0) * density_1 * mom_dot_ei * mom_dot_ei + 3.0 * mom_dot_ei);
				//------------------------------------------------------------------------------------------------------
				// Need to distinguish the int boundaryId = site.GetIoletId() correctly and pass the info (identify the proper ghost density and inlet-normals.
				//=============================================================================================================

				// printf("Site ID = %lld - Inlet in Dir: %d, Unstreamed direction: %d, fEq = %.5e \n\n", Ind, LB_Dir, unstreamed_dir, dev_fEq_unstr);

				// Case of NashZerothOrderPressure:
				// *latticeData->GetFNew(site.GetIndex() * LatticeType::NUMVECTORS + unstreamed) = ghostHydrovars.GetFEq()[unstreamed];
				GMem_dbl_fNew_b[(unsigned long long)unstreamed_dir * nArr_dbl + Ind] = dev_fEq_unstr;

			}
			else{ // bulkLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);

				// Use the Neighbouring Index given in GPUDataAddr_int64_Neigh_d, which is the actual streaming Array Index in f_new global memory
				int64_t dev_NeighInd = GMem_int64_Neigh[(unsigned long long)LB_Dir * nArr_dbl + Ind];

				// Save the post collision population in fNew
				GMem_dbl_fNew_b[dev_NeighInd] = dev_ff[LB_Dir];
				//
				// Debugging - Remove later
				// Check if it points to an address outside the (nFluid_nodes * _NUMVECTORS + 1+totalSharedFs )
				//if (dev_NeighInd[LB_Dir] >= (nArr_dbl*_NUMVECTORS+1+totalSharedFs)) printf("Error!!! Fluid Index = %lld, Stream.Dir.= %d, Max. Streaming addr = %lld Vs Stream. Addr.=%lld \n\n", Ind, LB_Dir, nArr_dbl*_NUMVECTORS+1+totalSharedFs, dev_NeighInd[LB_Dir] );
				//

				//---------------------------------------------------------------------------
			} // Closes the bulkLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);
		}
		//=============================================================================================


		// Write old density and velocity to memory -
		// Maybe use a different cuda kernel for these calculations (if saving the MacroVariables delays the collision/streaming kernel)
		// Check -  To do!!!
		if(time_Step%100==0){
			GMem_dbl_MacroVars[Ind] = nn;
			GMem_dbl_MacroVars[1ULL*nArr_dbl + Ind] = velx;
			GMem_dbl_MacroVars[2ULL*nArr_dbl + Ind] = vely;
			GMem_dbl_MacroVars[3ULL*nArr_dbl + Ind] = velz;
		}

	} // Ends the kernel GPU_Collide Type 2: mWallCollision: Case Fluid-Wall collision
	//==========================================================================================




	//**************************************************************
	// Kernel for the Collision step for the Lattice Boltzmann algorithm
	// 	Collision Type 3: mInletCollision: Inlet BCs - PreReceive
	//
	// Inlet BCs: specified with HEMELB_INLET_BOUNDARY in CMakeLists.txt
	//	Two Possible types of Inlet BCs:
	// 	1. NashZerothOrderPressure: Implement this first (see lb/streamers/NashZerothOrderPressureDelegate.h)
	//	2. LaddIolet: (see lb/streamers/LaddIoletDelegate.h)
	//
	// Implementation currently follows the memory arrangement of the data
	// by index LB, i.e. method (b)
	// Need to pass the information for the fluid-iolet links - Done!!!
	// This information is in ioletIntersection, see geometry/SiteDataBare.h
	//
	// This version uses the ACTUAL streaming address in global memory - NOT the fluid ID.
	//**************************************************************
	__global__ void GPU_CollideStream_3_NashZerothOrderPressure_Inlet_Inner(distribn_t* GMem_dbl_fOld_b,
																distribn_t* GMem_dbl_fNew_b,
																distribn_t* GMem_dbl_MacroVars,
																int64_t* GMem_int64_Neigh,
																uint32_t* GMem_uint32_Iolet_Link,
																distribn_t* GMem_ghostDensity,
																float* GMem_inletNormal,
																int nInlets,
																uint64_t nArr_dbl,
																uint64_t lower_limit, uint64_t upper_limit, uint64_t totalSharedFs, int time_Step, int num_local_Iolets)
	{
		unsigned long long Ind = blockIdx.x * blockDim.x + threadIdx.x;
		Ind = Ind + lower_limit;

		if(Ind >= upper_limit)
			return;

		// Load the distribution functions
		//f[19] and fEq[19]
		double dev_ff[19], dev_fEq[19];
		double nn = 0.0;	// density
		double momentum_x, momentum_y, momentum_z;
		momentum_x = momentum_y = momentum_z = 0.0;

		double velx, vely, velz;	// Fluid Velocity

		//-----------------------------------------------------------------------------------------------------------
		// 1. Read the fOld_GPU_b distr. functions
		// 2. Calculate the nessessary elements for calculating the equilibrium distribution functions
		// 		a. Calculate density
		// 		b. Calculate momentum - Needs to consider the case of body force as well - To do!!!
		for(int direction = 0; direction< _NUMVECTORS; direction++){
			dev_ff[direction] = GMem_dbl_fOld_b[(unsigned long long)direction * nArr_dbl + Ind];

			nn += dev_ff[direction];
			momentum_x += (double)_CX_19[direction] * dev_ff[direction];
			momentum_y += (double)_CY_19[direction] * dev_ff[direction];
			momentum_z += (double)_CZ_19[direction] * dev_ff[direction];
			//printf("Momentum: _x = %.5e, _y = %.5e, _z = %.5e \n\n", momentum_x, momentum_y, momentum_z);
		}


		// In the case of body force
		//momentum_x += 0.5 * _force_x;
		//momentum_y += 0.5 * _force_y;
		//momentum_z += 0.5 * _force_z;


		// Compute velocity components
		velx = momentum_x/nn;
		vely = momentum_y/nn;
		velz = momentum_z/nn;

		//-----------------------------------------------------------------------------------------------------------
		// c. Calculate equilibrium distr. functions
		double density_1 = 1.0 / nn;
		double momentumMagnitudeSquared = momentum_x * momentum_x
													+ momentum_y * momentum_y + momentum_z * momentum_z;

		for (int i = 0; i < _NUMVECTORS; ++i)
		{
			double mom_dot_ei = (double)_CX_19[i] * momentum_x
									+ (double)_CY_19[i] * momentum_y
									+ (double)_CZ_19[i] * momentum_z;

			dev_fEq[i] = _EQMWEIGHTS_19[i]
							* (nn - (3.0 / 2.0) * momentumMagnitudeSquared * density_1
											+ (9.0 / 2.0) * density_1 * mom_dot_ei * mom_dot_ei + 3.0 * mom_dot_ei);
		}
		//-----------------------------------------------------------------------------------------------------------

		// d. Body Force case: Add details of any forcing scheme here - Evaluate force[i]
		// To do!!!
		//-----------------------------------------------------------------------------------------------------------

		// Collision step:
		// Single Relaxation Time approximation (LBGK)
		//double dev_fn[19];		// or maybe use the existing dev_ff[_NUMVECTORS] to minimise the memory requirements - Check and replace in the future

		// Evolution equation for the fi's here
		for (int i = 0; i < _NUMVECTORS; ++i)
		{
			//dev_fn[i] = dev_ff[i] + (dev_fEq[i] - dev_ff[i])/dev_tau; // + force[i];
			dev_ff[i] += (dev_ff[i] - dev_fEq[i]) * dev_minusInvTau; // Check if multiplying by dev_minusInvTau makes a difference
		}
		//__syncthreads(); // Check if needed!


		// --------------------------------------------------------------------------------
		// Streaming Step:
		// a. Load the streaming indices
		// b. The Iolet-Fluid links info
		// c. The ghost density
		// d. The inletNormal

		// a. Bulk Streaming indices: dev_NeighInd[19] here refers to the ACTUAL Streaming Array index (Data Address) in f_old and f_new
		//int64_t dev_NeighInd[19];

		// printf("Number of inlets: %d \n\n", nInlets);
		distribn_t ghost_dens; // = 0.0; //new distribn_t[nInlets];	// c. The ghost density
		float inletNormal_x, inletNormal_y, inletNormal_z;

	/*
		for(int LB_Dir=0; LB_Dir< _NUMVECTORS; LB_Dir++){

			// If we use the elements in GMem_int64_Neigh - then we access the memory address in fOld or fNew directly (not the fluid id)
			// (remember the memory layout in hemeLB is based on the site fluid index, i.e. f0[0], f1[0], f2[0], ..., fq[0] and for the Fluid Index Ind : f0[Ind], f1[Ind], f2[Ind], ..., fq[Ind]

			dev_NeighInd[LB_Dir] = GMem_int64_Neigh[(unsigned long long)LB_Dir * nArr_dbl + Ind]; // Read the streaming info here - Here Refers to Data Address NOT THE STREAMING FLUID ID!!!
		}
		__syncthreads();
	*/
		//
		// b. Iolet-Fluid links info:
		uint32_t Iolet_Intersect = GMem_uint32_Iolet_Link[Ind];


		// Read the ghost density and the inlet Normal
		// How do I distinguish which inlet ID do I have ??? Need to think about this... To do!!!
		// Need to pass this info based on the site Index (from the initialisation process. With given site ranges -> int boundaryId = site.GetIoletId();)

		// Access the info from the constant memory: _Iolets_Inlet_Inner[local_iolets_MaxSIZE], local_iolets_MaxSIZE = 6 cuda_params.h (Assume 2 max iolets per RANK)
		// Determine the IdInlet - Done!!!
		int IdInlet = INT32_MAX; // Iolet (Inlet/Outlet) ID
		if(num_local_Iolets==1){
			IdInlet = _Iolets_Inlet_Inner[0];// IdInlet = iolets_ID_range[0];
		}
		else{
			// Call a device function to determine which is the Iolet ID - using the iolets_ID_range Array
			// iolets_ID_range Array:
			//	a. Size: num_local_Iolets * 3
			// 	b. Iolet ID, Range of fluid IDs: [lower_limit, upper_limit)
			_determine_Iolet_ID(num_local_Iolets, _Iolets_Inlet_Inner, Ind, &IdInlet); // _determine_Iolet_ID(num_local_Iolets, iolets_ID_range, Ind, &IdInlet);
		}

		// Testing:
		if(IdInlet==INT32_MAX)
		{
			printf("Fluid_ID : %lld, ID_iolet: %d - Fluid NOT in IOLET range!!! \n\n", Ind, IdInlet);
		}
		/*else{
			printf("Fluid_ID : %lld, ID_iolet: %d \n\n", Ind, IdInlet);
		}
		*/

		//printf("Number of local Iolets = %d \n", num_local_Iolets);
/*
		for (int index = 0; index < num_local_Iolets; index++){
			printf(" Iolet ID: %d, lower_range: %lld, upper_range: %lld ", _Iolets_Inlet_Inner[3*index], _Iolets_Inlet_Inner[3*index+1], _Iolets_Inlet_Inner[3*index+2]);
		}
		printf("\n\n");
*/

		ghost_dens = GMem_ghostDensity[IdInlet];
		inletNormal_x = GMem_inletNormal[3*IdInlet];
		inletNormal_y = GMem_inletNormal[3*IdInlet+1];
		inletNormal_z = GMem_inletNormal[3*IdInlet+2];
	//		printf("ghost_dens[%d]: %.5f, inletNormal_x = %.5f, inletNormal_y = %.5f, inletNormal_z = %.5f  \n\n", IdInlet, ghost_dens, inletNormal_x, inletNormal_y, inletNormal_z);


		// Put the new populations after collision in the GMem_dbl array,
		// implementing the streaming step with Simple Bounce Back if Wall-Fluid link

		// fNew (dev_fn) populations:
		for (int LB_Dir = 0; LB_Dir < _NUMVECTORS; LB_Dir++)
		{
			unsigned mask = 1U << (LB_Dir - 1); // Needs to left shift the bits in mask so that I can then compare against the value in test_Wall_Intersect (To do: compare against test_bool_Wall_Intersect as well)
			bool is_Iolet_link = (Iolet_Intersect & mask);

			if(is_Iolet_link){	// ioletLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);

				//=============================================================================================================
				// Not valid in general! Need to change!!!
				// (IdInlet=0) Here we assume that we have only one inlet and the value of int boundaryId = site.GetIoletId() = 1. Need to change in the future!!!
				double component = velx*inletNormal_x + vely*inletNormal_y + velz*inletNormal_z;	// distribn_t component = (hydroVars.momentum / hydroVars.density).Dot(ioletNormal);

				// ghostHydrovars.momentum = ioletNormal * component * ghostDensity;
				double momentum_x = inletNormal_x * component * ghost_dens;
				double momentum_y = inletNormal_y * component * ghost_dens;
				double momentum_z = inletNormal_z * component * ghost_dens;


				//------------------------------------------------------------------------------------------------------
				// Calculate Feq[unstreamed_dir] - Only the direction that is necessary
				density_1 = 1.0 / ghost_dens;
				momentumMagnitudeSquared = momentum_x * momentum_x
													+ momentum_y * momentum_y + momentum_z * momentum_z;

				int unstreamed_dir = _InvDirections_19[LB_Dir];
				double mom_dot_ei = (double)_CX_19[unstreamed_dir] * momentum_x
									+ (double)_CY_19[unstreamed_dir] * momentum_y
									+ (double)_CZ_19[unstreamed_dir] * momentum_z;

				double dev_fEq_unstr = _EQMWEIGHTS_19[unstreamed_dir]
							* (ghost_dens - (3.0 / 2.0) * momentumMagnitudeSquared * density_1
											+ (9.0 / 2.0) * density_1 * mom_dot_ei * mom_dot_ei + 3.0 * mom_dot_ei);
				//------------------------------------------------------------------------------------------------------
				// Need to distinguish the int boundaryId = site.GetIoletId() correctly and pass the info (identify the proper ghost density and inlet-normals.
				//=============================================================================================================

				// printf("Site ID = %lld - Inlet in Dir: %d, Unstreamed direction: %d, fEq = %.5e \n\n", Ind, LB_Dir, unstreamed_dir, dev_fEq_unstr);

				// Case of NashZerothOrderPressure:
				// *latticeData->GetFNew(site.GetIndex() * LatticeType::NUMVECTORS + unstreamed) = ghostHydrovars.GetFEq()[unstreamed];
				GMem_dbl_fNew_b[(unsigned long long)unstreamed_dir * nArr_dbl + Ind] = dev_fEq_unstr;

			}
			else{ // bulkLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);

				// Use the Neighbouring Index given in GPUDataAddr_int64_Neigh_d, which is the actual streaming Array Index in f_new global memory
				int64_t dev_NeighInd = GMem_int64_Neigh[(unsigned long long)LB_Dir * nArr_dbl + Ind];

				// Save the post collision population in fNew
				GMem_dbl_fNew_b[dev_NeighInd] = dev_ff[LB_Dir];
				//
				// Debugging - Remove later
				// Check if it points to an address outside the (nFluid_nodes * _NUMVECTORS + 1+totalSharedFs )
				//if (dev_NeighInd[LB_Dir] >= (nArr_dbl*_NUMVECTORS+1+totalSharedFs)) printf("Error!!! Fluid Index = %lld, Stream.Dir.= %d, Max. Streaming addr = %lld Vs Stream. Addr.=%lld \n\n", Ind, LB_Dir, nArr_dbl*_NUMVECTORS+1+totalSharedFs, dev_NeighInd[LB_Dir] );
				//

				//---------------------------------------------------------------------------
			} // Closes the bulkLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);
		}
		//=============================================================================================


		// Write old density and velocity to memory -
		// Maybe use a different cuda kernel for these calculations (if saving the MacroVariables delays the collision/streaming kernel)
		// Check -  To do!!!
		if(time_Step%100==0){
			GMem_dbl_MacroVars[Ind] = nn;
			GMem_dbl_MacroVars[1ULL*nArr_dbl + Ind] = velx;
			GMem_dbl_MacroVars[2ULL*nArr_dbl + Ind] = vely;
			GMem_dbl_MacroVars[3ULL*nArr_dbl + Ind] = velz;
		}

	} // Ends the kernel GPU_Collide Type 3: mInletCollision
	//==========================================================================================



	//**************************************************************
	// Kernel for the Collision step for the Lattice Boltzmann algorithm
	// 	Collision Type 3: mInletCollision: Inlet BCs - PreSend
	//
	// Inlet BCs: specified with HEMELB_INLET_BOUNDARY in CMakeLists.txt
	//	Two Possible types of Inlet BCs:
	// 	1. NashZerothOrderPressure: Implement this first (see lb/streamers/NashZerothOrderPressureDelegate.h)
	//	2. LaddIolet: (see lb/streamers/LaddIoletDelegate.h)
	//
	// Implementation currently follows the memory arrangement of the data
	// by index LB, i.e. method (b)
	// Need to pass the information for the fluid-iolet links - Done!!!
	// This information is in ioletIntersection, see geometry/SiteDataBare.h
	//
	// This version uses the ACTUAL streaming address in global memory - NOT the fluid ID.
	//**************************************************************
	__global__ void GPU_CollideStream_3_NashZerothOrderPressure_Inlet_Edge(distribn_t* GMem_dbl_fOld_b,
																distribn_t* GMem_dbl_fNew_b,
																distribn_t* GMem_dbl_MacroVars,
																int64_t* GMem_int64_Neigh,
																uint32_t* GMem_uint32_Iolet_Link,
																distribn_t* GMem_ghostDensity,
																float* GMem_inletNormal,
																int nInlets,
																uint64_t nArr_dbl,
																uint64_t lower_limit, uint64_t upper_limit, uint64_t totalSharedFs, int time_Step, int num_local_Iolets)
	{
		unsigned long long Ind = blockIdx.x * blockDim.x + threadIdx.x;
		Ind = Ind + lower_limit;

		if(Ind >= upper_limit)
			return;

		// Load the distribution functions
		//f[19] and fEq[19]
		double dev_ff[19], dev_fEq[19];
		double nn = 0.0;	// density
		double momentum_x, momentum_y, momentum_z;
		momentum_x = momentum_y = momentum_z = 0.0;

		double velx, vely, velz;	// Fluid Velocity

		//-----------------------------------------------------------------------------------------------------------
		// 1. Read the fOld_GPU_b distr. functions
		// 2. Calculate the nessessary elements for calculating the equilibrium distribution functions
		// 		a. Calculate density
		// 		b. Calculate momentum - Needs to consider the case of body force as well - To do!!!
		for(int direction = 0; direction< _NUMVECTORS; direction++){
			dev_ff[direction] = GMem_dbl_fOld_b[(unsigned long long)direction * nArr_dbl + Ind];

			nn += dev_ff[direction];
			momentum_x += (double)_CX_19[direction] * dev_ff[direction];
			momentum_y += (double)_CY_19[direction] * dev_ff[direction];
			momentum_z += (double)_CZ_19[direction] * dev_ff[direction];
			//printf("Momentum: _x = %.5e, _y = %.5e, _z = %.5e \n\n", momentum_x, momentum_y, momentum_z);
		}


		// In the case of body force
		//momentum_x += 0.5 * _force_x;
		//momentum_y += 0.5 * _force_y;
		//momentum_z += 0.5 * _force_z;


		// Compute velocity components
		velx = momentum_x/nn;
		vely = momentum_y/nn;
		velz = momentum_z/nn;

		//-----------------------------------------------------------------------------------------------------------
		// c. Calculate equilibrium distr. functions
		double density_1 = 1.0 / nn;
		double momentumMagnitudeSquared = momentum_x * momentum_x
													+ momentum_y * momentum_y + momentum_z * momentum_z;

		for (int i = 0; i < _NUMVECTORS; ++i)
		{
			double mom_dot_ei = (double)_CX_19[i] * momentum_x
									+ (double)_CY_19[i] * momentum_y
									+ (double)_CZ_19[i] * momentum_z;

			dev_fEq[i] = _EQMWEIGHTS_19[i]
							* (nn - (3.0 / 2.0) * momentumMagnitudeSquared * density_1
											+ (9.0 / 2.0) * density_1 * mom_dot_ei * mom_dot_ei + 3.0 * mom_dot_ei);
		}
		//-----------------------------------------------------------------------------------------------------------

		// d. Body Force case: Add details of any forcing scheme here - Evaluate force[i]
		// To do!!!
		//-----------------------------------------------------------------------------------------------------------

		// Collision step:
		// Single Relaxation Time approximation (LBGK)
		//double dev_fn[19];		// or maybe use the existing dev_ff[_NUMVECTORS] to minimise the memory requirements - Check and replace in the future

		// Evolution equation for the fi's here
		for (int i = 0; i < _NUMVECTORS; ++i)
		{
			//dev_fn[i] = dev_ff[i] + (dev_fEq[i] - dev_ff[i])/dev_tau; // + force[i];
			dev_ff[i] += (dev_ff[i] - dev_fEq[i]) * dev_minusInvTau; // Check if multiplying by dev_minusInvTau makes a difference
		}
		//__syncthreads(); // Check if needed!


		// --------------------------------------------------------------------------------
		// Streaming Step:
		// a. Load the streaming indices
		// b. The Iolet-Fluid links info
		// c. The ghost density
		// d. The inletNormal

		// a. Bulk Streaming indices: dev_NeighInd[19] here refers to the ACTUAL Streaming Array index (Data Address) in f_old and f_new
		//int64_t dev_NeighInd[19];

		// printf("Number of inlets: %d \n\n", nInlets);
		distribn_t ghost_dens; // = 0.0; //new distribn_t[nInlets];	// c. The ghost density
		float inletNormal_x, inletNormal_y, inletNormal_z;

	/*
		for(int LB_Dir=0; LB_Dir< _NUMVECTORS; LB_Dir++){

			// If we use the elements in GMem_int64_Neigh - then we access the memory address in fOld or fNew directly (not the fluid id)
			// (remember the memory layout in hemeLB is based on the site fluid index, i.e. f0[0], f1[0], f2[0], ..., fq[0] and for the Fluid Index Ind : f0[Ind], f1[Ind], f2[Ind], ..., fq[Ind]

			dev_NeighInd[LB_Dir] = GMem_int64_Neigh[(unsigned long long)LB_Dir * nArr_dbl + Ind]; // Read the streaming info here - Here Refers to Data Address NOT THE STREAMING FLUID ID!!!
		}
		__syncthreads();
	*/
		//
		// b. Iolet-Fluid links info:
		uint32_t Iolet_Intersect = GMem_uint32_Iolet_Link[Ind];


		// Read the ghost density and the inlet Normal
		// How do I distinguish which inlet ID do I have ??? Need to think about this... To do!!!
		// Need to pass this info based on the site Index (from the initialisation process. With given site ranges -> int boundaryId = site.GetIoletId();)

		// Access the info from the constant memory: _Iolets_Inlet_Inner[local_iolets_MaxSIZE], local_iolets_MaxSIZE = 6 cuda_params.h (Assume 2 max iolets per RANK)
		// Determine the IdInlet - Done!!!
		int IdInlet = INT32_MAX; // Iolet (Inlet/Outlet) ID
		if(num_local_Iolets==1){
			IdInlet = _Iolets_Inlet_Edge[0];// IdInlet = iolets_ID_range[0];
		}
		else{
			// Call a device function to determine which is the Iolet ID - using the iolets_ID_range Array
			// iolets_ID_range Array:
			//	a. Size: num_local_Iolets * 3
			// 	b. Iolet ID, Range of fluid IDs: [lower_limit, upper_limit)
			_determine_Iolet_ID(num_local_Iolets, _Iolets_Inlet_Edge, Ind, &IdInlet); // _determine_Iolet_ID(num_local_Iolets, iolets_ID_range, Ind, &IdInlet);
		}

		// Testing:
		if(IdInlet==INT32_MAX)
		{
			printf("Fluid_ID : %lld, ID_iolet: %d - Fluid NOT in IOLET range!!! \n\n", Ind, IdInlet);
		}
		/*else{
			printf("Fluid_ID : %lld, ID_iolet: %d \n\n", Ind, IdInlet);
		}
		*/

		//printf("Number of local Iolets = %d \n", num_local_Iolets);
/*
		for (int index = 0; index < num_local_Iolets; index++){
			printf(" Iolet ID: %d, lower_range: %lld, upper_range: %lld ", _Iolets_Inlet_Inner[3*index], _Iolets_Inlet_Inner[3*index+1], _Iolets_Inlet_Inner[3*index+2]);
		}
		printf("\n\n");
*/

		ghost_dens = GMem_ghostDensity[IdInlet];
		inletNormal_x = GMem_inletNormal[3*IdInlet];
		inletNormal_y = GMem_inletNormal[3*IdInlet+1];
		inletNormal_z = GMem_inletNormal[3*IdInlet+2];
	//		printf("ghost_dens[%d]: %.5f, inletNormal_x = %.5f, inletNormal_y = %.5f, inletNormal_z = %.5f  \n\n", IdInlet, ghost_dens, inletNormal_x, inletNormal_y, inletNormal_z);


		// Put the new populations after collision in the GMem_dbl array,
		// implementing the streaming step with Simple Bounce Back if Wall-Fluid link

		// fNew (dev_fn) populations:
		for (int LB_Dir = 0; LB_Dir < _NUMVECTORS; LB_Dir++)
		{
			unsigned mask = 1U << (LB_Dir - 1); // Needs to left shift the bits in mask so that I can then compare against the value in test_Wall_Intersect (To do: compare against test_bool_Wall_Intersect as well)
			bool is_Iolet_link = (Iolet_Intersect & mask);

			if(is_Iolet_link){	// ioletLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);

				//=============================================================================================================
				// Not valid in general! Need to change!!!
				// (IdInlet=0) Here we assume that we have only one inlet and the value of int boundaryId = site.GetIoletId() = 1. Need to change in the future!!!
				double component = velx*inletNormal_x + vely*inletNormal_y + velz*inletNormal_z;	// distribn_t component = (hydroVars.momentum / hydroVars.density).Dot(ioletNormal);

				// ghostHydrovars.momentum = ioletNormal * component * ghostDensity;
				double momentum_x = inletNormal_x * component * ghost_dens;
				double momentum_y = inletNormal_y * component * ghost_dens;
				double momentum_z = inletNormal_z * component * ghost_dens;


				//------------------------------------------------------------------------------------------------------
				// Calculate Feq[unstreamed_dir] - Only the direction that is necessary
				density_1 = 1.0 / ghost_dens;
				momentumMagnitudeSquared = momentum_x * momentum_x
													+ momentum_y * momentum_y + momentum_z * momentum_z;

				int unstreamed_dir = _InvDirections_19[LB_Dir];
				double mom_dot_ei = (double)_CX_19[unstreamed_dir] * momentum_x
									+ (double)_CY_19[unstreamed_dir] * momentum_y
									+ (double)_CZ_19[unstreamed_dir] * momentum_z;

				double dev_fEq_unstr = _EQMWEIGHTS_19[unstreamed_dir]
							* (ghost_dens - (3.0 / 2.0) * momentumMagnitudeSquared * density_1
											+ (9.0 / 2.0) * density_1 * mom_dot_ei * mom_dot_ei + 3.0 * mom_dot_ei);
				//------------------------------------------------------------------------------------------------------
				// Need to distinguish the int boundaryId = site.GetIoletId() correctly and pass the info (identify the proper ghost density and inlet-normals.
				//=============================================================================================================

				// printf("Site ID = %lld - Inlet in Dir: %d, Unstreamed direction: %d, fEq = %.5e \n\n", Ind, LB_Dir, unstreamed_dir, dev_fEq_unstr);

				// Case of NashZerothOrderPressure:
				// *latticeData->GetFNew(site.GetIndex() * LatticeType::NUMVECTORS + unstreamed) = ghostHydrovars.GetFEq()[unstreamed];
				GMem_dbl_fNew_b[(unsigned long long)unstreamed_dir * nArr_dbl + Ind] = dev_fEq_unstr;

			}
			else{ // bulkLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);

				// Use the Neighbouring Index given in GPUDataAddr_int64_Neigh_d, which is the actual streaming Array Index in f_new global memory
				int64_t dev_NeighInd = GMem_int64_Neigh[(unsigned long long)LB_Dir * nArr_dbl + Ind];

				// Save the post collision population in fNew
				GMem_dbl_fNew_b[dev_NeighInd] = dev_ff[LB_Dir];
				//
				// Debugging - Remove later
				// Check if it points to an address outside the (nFluid_nodes * _NUMVECTORS + 1+totalSharedFs )
				//if (dev_NeighInd[LB_Dir] >= (nArr_dbl*_NUMVECTORS+1+totalSharedFs)) printf("Error!!! Fluid Index = %lld, Stream.Dir.= %d, Max. Streaming addr = %lld Vs Stream. Addr.=%lld \n\n", Ind, LB_Dir, nArr_dbl*_NUMVECTORS+1+totalSharedFs, dev_NeighInd[LB_Dir] );
				//

				//---------------------------------------------------------------------------
			} // Closes the bulkLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);
		}
		//=============================================================================================


		// Write old density and velocity to memory -
		// Maybe use a different cuda kernel for these calculations (if saving the MacroVariables delays the collision/streaming kernel)
		// Check -  To do!!!
		if(time_Step%100==0){
			GMem_dbl_MacroVars[Ind] = nn;
			GMem_dbl_MacroVars[1ULL*nArr_dbl + Ind] = velx;
			GMem_dbl_MacroVars[2ULL*nArr_dbl + Ind] = vely;
			GMem_dbl_MacroVars[3ULL*nArr_dbl + Ind] = velz;
		}

	} // Ends the kernel GPU_Collide Type 3: mInletCollision - PreSend
	//==========================================================================================




	//**************************************************************
	// Kernel for the Collision step for the Lattice Boltzmann algorithm
	// 	Collision Type 4: mOutletCollision: Outlet BCs - PreReceive
	//
	// Inlet BCs: specified with HEMELB_INLET_BOUNDARY in CMakeLists.txt
	//	Two Possible types of Inlet BCs:
	// 	1. NashZerothOrderPressure: Implement this first (see lb/streamers/NashZerothOrderPressureDelegate.h)
	//	2. LaddIolet: (see lb/streamers/LaddIoletDelegate.h)
	//
	// Implementation currently follows the memory arrangement of the data
	// by index LB, i.e. method (b)
	// Need to pass the information for the fluid-iolet links - Done!!!
	// This information is in ioletIntersection, see geometry/SiteDataBare.h
	//
	// This version uses the ACTUAL streaming address in global memory - NOT the fluid ID.
	//**************************************************************
	__global__ void GPU_CollideStream_3_NashZerothOrderPressure_Outlet_Inner(distribn_t* GMem_dbl_fOld_b,
																distribn_t* GMem_dbl_fNew_b,
																distribn_t* GMem_dbl_MacroVars,
																int64_t* GMem_int64_Neigh,
																uint32_t* GMem_uint32_Iolet_Link,
																distribn_t* GMem_ghostDensity,
																float* GMem_inletNormal,
																int nInlets,
																uint64_t nArr_dbl,
																uint64_t lower_limit, uint64_t upper_limit, uint64_t totalSharedFs, int time_Step, int num_local_Iolets)
	{
		unsigned long long Ind = blockIdx.x * blockDim.x + threadIdx.x;
		Ind = Ind + lower_limit;

		if(Ind >= upper_limit)
			return;

		// Load the distribution functions
		//f[19] and fEq[19]
		double dev_ff[19], dev_fEq[19];
		double nn = 0.0;	// density
		double momentum_x, momentum_y, momentum_z;
		momentum_x = momentum_y = momentum_z = 0.0;

		double velx, vely, velz;	// Fluid Velocity

		//-----------------------------------------------------------------------------------------------------------
		// 1. Read the fOld_GPU_b distr. functions
		// 2. Calculate the nessessary elements for calculating the equilibrium distribution functions
		// 		a. Calculate density
		// 		b. Calculate momentum - Needs to consider the case of body force as well - To do!!!
		for(int direction = 0; direction< _NUMVECTORS; direction++){
			dev_ff[direction] = GMem_dbl_fOld_b[(unsigned long long)direction * nArr_dbl + Ind];

			nn += dev_ff[direction];
			momentum_x += (double)_CX_19[direction] * dev_ff[direction];
			momentum_y += (double)_CY_19[direction] * dev_ff[direction];
			momentum_z += (double)_CZ_19[direction] * dev_ff[direction];
			//printf("Momentum: _x = %.5e, _y = %.5e, _z = %.5e \n\n", momentum_x, momentum_y, momentum_z);
		}


		// In the case of body force
		//momentum_x += 0.5 * _force_x;
		//momentum_y += 0.5 * _force_y;
		//momentum_z += 0.5 * _force_z;


		// Compute velocity components
		velx = momentum_x/nn;
		vely = momentum_y/nn;
		velz = momentum_z/nn;

		//-----------------------------------------------------------------------------------------------------------
		// c. Calculate equilibrium distr. functions
		double density_1 = 1.0 / nn;
		double momentumMagnitudeSquared = momentum_x * momentum_x
													+ momentum_y * momentum_y + momentum_z * momentum_z;

		for (int i = 0; i < _NUMVECTORS; ++i)
		{
			double mom_dot_ei = (double)_CX_19[i] * momentum_x
									+ (double)_CY_19[i] * momentum_y
									+ (double)_CZ_19[i] * momentum_z;

			dev_fEq[i] = _EQMWEIGHTS_19[i]
							* (nn - (3.0 / 2.0) * momentumMagnitudeSquared * density_1
											+ (9.0 / 2.0) * density_1 * mom_dot_ei * mom_dot_ei + 3.0 * mom_dot_ei);
		}
		//-----------------------------------------------------------------------------------------------------------

		// d. Body Force case: Add details of any forcing scheme here - Evaluate force[i]
		// To do!!!
		//-----------------------------------------------------------------------------------------------------------

		// Collision step:
		// Single Relaxation Time approximation (LBGK)
		//double dev_fn[19];		// or maybe use the existing dev_ff[_NUMVECTORS] to minimise the memory requirements - Check and replace in the future

		// Evolution equation for the fi's here
		for (int i = 0; i < _NUMVECTORS; ++i)
		{
			//dev_fn[i] = dev_ff[i] + (dev_fEq[i] - dev_ff[i])/dev_tau; // + force[i];
			dev_ff[i] += (dev_ff[i] - dev_fEq[i]) * dev_minusInvTau; // Check if multiplying by dev_minusInvTau makes a difference
		}
		//__syncthreads(); // Check if needed!


		// --------------------------------------------------------------------------------
		// Streaming Step:
		// a. Load the streaming indices
		// b. The Iolet-Fluid links info
		// c. The ghost density
		// d. The inletNormal

		// a. Bulk Streaming indices: dev_NeighInd[19] here refers to the ACTUAL Streaming Array index (Data Address) in f_old and f_new
		//int64_t dev_NeighInd[19];

		// printf("Number of inlets: %d \n\n", nInlets);
		distribn_t ghost_dens; // = 0.0; //new distribn_t[nInlets];	// c. The ghost density
		float inletNormal_x, inletNormal_y, inletNormal_z;

	/*
		for(int LB_Dir=0; LB_Dir< _NUMVECTORS; LB_Dir++){

			// If we use the elements in GMem_int64_Neigh - then we access the memory address in fOld or fNew directly (not the fluid id)
			// (remember the memory layout in hemeLB is based on the site fluid index, i.e. f0[0], f1[0], f2[0], ..., fq[0] and for the Fluid Index Ind : f0[Ind], f1[Ind], f2[Ind], ..., fq[Ind]

			dev_NeighInd[LB_Dir] = GMem_int64_Neigh[(unsigned long long)LB_Dir * nArr_dbl + Ind]; // Read the streaming info here - Here Refers to Data Address NOT THE STREAMING FLUID ID!!!
		}
		__syncthreads();
	*/
		//
		// b. Iolet-Fluid links info:
		uint32_t Iolet_Intersect = GMem_uint32_Iolet_Link[Ind];


		// Read the ghost density and the inlet Normal
		// How do I distinguish which inlet ID do I have ??? Need to think about this... To do!!!
		// Need to pass this info based on the site Index (from the initialisation process. With given site ranges -> int boundaryId = site.GetIoletId();)

		// Access the info from the constant memory: _Iolets_Inlet_Inner[local_iolets_MaxSIZE], local_iolets_MaxSIZE = 6 cuda_params.h (Assume 2 max iolets per RANK)
		// Determine the IdInlet - Done!!!
		int IdInlet = INT32_MAX; // Iolet (Inlet/Outlet) ID
		if(num_local_Iolets==1){
			IdInlet = _Iolets_Outlet_Inner[0];// IdInlet = iolets_ID_range[0];
		}
		else{
			// Call a device function to determine which is the Iolet ID - using the iolets_ID_range Array
			// iolets_ID_range Array:
			//	a. Size: num_local_Iolets * 3
			// 	b. Iolet ID, Range of fluid IDs: [lower_limit, upper_limit)
			_determine_Iolet_ID(num_local_Iolets, _Iolets_Outlet_Inner, Ind, &IdInlet); // _determine_Iolet_ID(num_local_Iolets, iolets_ID_range, Ind, &IdInlet);
		}

		// Testing:
		if(IdInlet==INT32_MAX)
		{
			printf("Fluid_ID : %lld, ID_iolet: %d - Fluid NOT in IOLET range!!! \n\n", Ind, IdInlet);
		}
		/*else{
			printf("Fluid_ID : %lld, ID_iolet: %d \n\n", Ind, IdInlet);
		}
		*/

		//printf("Number of local Iolets = %d \n", num_local_Iolets);
/*
		for (int index = 0; index < num_local_Iolets; index++){
			printf(" Iolet ID: %d, lower_range: %lld, upper_range: %lld ", _Iolets_Inlet_Inner[3*index], _Iolets_Inlet_Inner[3*index+1], _Iolets_Inlet_Inner[3*index+2]);
		}
		printf("\n\n");
*/

		ghost_dens = GMem_ghostDensity[IdInlet];
		inletNormal_x = GMem_inletNormal[3*IdInlet];
		inletNormal_y = GMem_inletNormal[3*IdInlet+1];
		inletNormal_z = GMem_inletNormal[3*IdInlet+2];
	//		printf("ghost_dens[%d]: %.5f, inletNormal_x = %.5f, inletNormal_y = %.5f, inletNormal_z = %.5f  \n\n", IdInlet, ghost_dens, inletNormal_x, inletNormal_y, inletNormal_z);


		// Put the new populations after collision in the GMem_dbl array,
		// implementing the streaming step with Simple Bounce Back if Wall-Fluid link

		// fNew (dev_fn) populations:
		for (int LB_Dir = 0; LB_Dir < _NUMVECTORS; LB_Dir++)
		{
			unsigned mask = 1U << (LB_Dir - 1); // Needs to left shift the bits in mask so that I can then compare against the value in test_Wall_Intersect (To do: compare against test_bool_Wall_Intersect as well)
			bool is_Iolet_link = (Iolet_Intersect & mask);

			if(is_Iolet_link){	// ioletLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);

				//=============================================================================================================
				// Not valid in general! Need to change!!!
				// (IdInlet=0) Here we assume that we have only one inlet and the value of int boundaryId = site.GetIoletId() = 1. Need to change in the future!!!
				double component = velx*inletNormal_x + vely*inletNormal_y + velz*inletNormal_z;	// distribn_t component = (hydroVars.momentum / hydroVars.density).Dot(ioletNormal);

				// ghostHydrovars.momentum = ioletNormal * component * ghostDensity;
				double momentum_x = inletNormal_x * component * ghost_dens;
				double momentum_y = inletNormal_y * component * ghost_dens;
				double momentum_z = inletNormal_z * component * ghost_dens;


				//------------------------------------------------------------------------------------------------------
				// Calculate Feq[unstreamed_dir] - Only the direction that is necessary
				density_1 = 1.0 / ghost_dens;
				momentumMagnitudeSquared = momentum_x * momentum_x
													+ momentum_y * momentum_y + momentum_z * momentum_z;

				int unstreamed_dir = _InvDirections_19[LB_Dir];
				double mom_dot_ei = (double)_CX_19[unstreamed_dir] * momentum_x
									+ (double)_CY_19[unstreamed_dir] * momentum_y
									+ (double)_CZ_19[unstreamed_dir] * momentum_z;

				double dev_fEq_unstr = _EQMWEIGHTS_19[unstreamed_dir]
							* (ghost_dens - (3.0 / 2.0) * momentumMagnitudeSquared * density_1
											+ (9.0 / 2.0) * density_1 * mom_dot_ei * mom_dot_ei + 3.0 * mom_dot_ei);
				//------------------------------------------------------------------------------------------------------
				// Need to distinguish the int boundaryId = site.GetIoletId() correctly and pass the info (identify the proper ghost density and inlet-normals.
				//=============================================================================================================

				// printf("Site ID = %lld - Inlet in Dir: %d, Unstreamed direction: %d, fEq = %.5e \n\n", Ind, LB_Dir, unstreamed_dir, dev_fEq_unstr);

				// Case of NashZerothOrderPressure:
				// *latticeData->GetFNew(site.GetIndex() * LatticeType::NUMVECTORS + unstreamed) = ghostHydrovars.GetFEq()[unstreamed];
				GMem_dbl_fNew_b[(unsigned long long)unstreamed_dir * nArr_dbl + Ind] = dev_fEq_unstr;

			}
			else{ // bulkLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);

				// Use the Neighbouring Index given in GPUDataAddr_int64_Neigh_d, which is the actual streaming Array Index in f_new global memory
				int64_t dev_NeighInd = GMem_int64_Neigh[(unsigned long long)LB_Dir * nArr_dbl + Ind];

				// Save the post collision population in fNew
				GMem_dbl_fNew_b[dev_NeighInd] = dev_ff[LB_Dir];
				//
				// Debugging - Remove later
				// Check if it points to an address outside the (nFluid_nodes * _NUMVECTORS + 1+totalSharedFs )
				//if (dev_NeighInd[LB_Dir] >= (nArr_dbl*_NUMVECTORS+1+totalSharedFs)) printf("Error!!! Fluid Index = %lld, Stream.Dir.= %d, Max. Streaming addr = %lld Vs Stream. Addr.=%lld \n\n", Ind, LB_Dir, nArr_dbl*_NUMVECTORS+1+totalSharedFs, dev_NeighInd[LB_Dir] );
				//

				//---------------------------------------------------------------------------
			} // Closes the bulkLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);
		}
		//=============================================================================================


		// Write old density and velocity to memory -
		// Maybe use a different cuda kernel for these calculations (if saving the MacroVariables delays the collision/streaming kernel)
		// Check -  To do!!!
		if(time_Step%100==0){
			GMem_dbl_MacroVars[Ind] = nn;
			GMem_dbl_MacroVars[1ULL*nArr_dbl + Ind] = velx;
			GMem_dbl_MacroVars[2ULL*nArr_dbl + Ind] = vely;
			GMem_dbl_MacroVars[3ULL*nArr_dbl + Ind] = velz;
		}

	} // Ends the kernel GPU_Collide Type 4: mOutletCollision - PreReceive
	//==========================================================================================


	//**************************************************************
	// Kernel for the Collision step for the Lattice Boltzmann algorithm
	// 	Collision Type 4: mOutletCollision: Outlet BCs - PreSend
	//
	// Inlet BCs: specified with HEMELB_INLET_BOUNDARY in CMakeLists.txt
	//	Two Possible types of Inlet BCs:
	// 	1. NashZerothOrderPressure: Implement this first (see lb/streamers/NashZerothOrderPressureDelegate.h)
	//	2. LaddIolet: (see lb/streamers/LaddIoletDelegate.h)
	//
	// Implementation currently follows the memory arrangement of the data
	// by index LB, i.e. method (b)
	// Need to pass the information for the fluid-iolet links - Done!!!
	// This information is in ioletIntersection, see geometry/SiteDataBare.h
	//
	// This version uses the ACTUAL streaming address in global memory - NOT the fluid ID.
	//**************************************************************
	__global__ void GPU_CollideStream_3_NashZerothOrderPressure_Outlet_Edge(distribn_t* GMem_dbl_fOld_b,
																distribn_t* GMem_dbl_fNew_b,
																distribn_t* GMem_dbl_MacroVars,
																int64_t* GMem_int64_Neigh,
																uint32_t* GMem_uint32_Iolet_Link,
																distribn_t* GMem_ghostDensity,
																float* GMem_inletNormal,
																int nInlets,
																uint64_t nArr_dbl,
																uint64_t lower_limit, uint64_t upper_limit, uint64_t totalSharedFs, int time_Step, int num_local_Iolets)
	{
		unsigned long long Ind = blockIdx.x * blockDim.x + threadIdx.x;
		Ind = Ind + lower_limit;

		if(Ind >= upper_limit)
			return;

		// Load the distribution functions
		//f[19] and fEq[19]
		double dev_ff[19], dev_fEq[19];
		double nn = 0.0;	// density
		double momentum_x, momentum_y, momentum_z;
		momentum_x = momentum_y = momentum_z = 0.0;

		double velx, vely, velz;	// Fluid Velocity

		//-----------------------------------------------------------------------------------------------------------
		// 1. Read the fOld_GPU_b distr. functions
		// 2. Calculate the nessessary elements for calculating the equilibrium distribution functions
		// 		a. Calculate density
		// 		b. Calculate momentum - Needs to consider the case of body force as well - To do!!!
		for(int direction = 0; direction< _NUMVECTORS; direction++){
			dev_ff[direction] = GMem_dbl_fOld_b[(unsigned long long)direction * nArr_dbl + Ind];

			nn += dev_ff[direction];
			momentum_x += (double)_CX_19[direction] * dev_ff[direction];
			momentum_y += (double)_CY_19[direction] * dev_ff[direction];
			momentum_z += (double)_CZ_19[direction] * dev_ff[direction];
			//printf("Momentum: _x = %.5e, _y = %.5e, _z = %.5e \n\n", momentum_x, momentum_y, momentum_z);
		}


		// In the case of body force
		//momentum_x += 0.5 * _force_x;
		//momentum_y += 0.5 * _force_y;
		//momentum_z += 0.5 * _force_z;


		// Compute velocity components
		velx = momentum_x/nn;
		vely = momentum_y/nn;
		velz = momentum_z/nn;

		//-----------------------------------------------------------------------------------------------------------
		// c. Calculate equilibrium distr. functions
		double density_1 = 1.0 / nn;
		double momentumMagnitudeSquared = momentum_x * momentum_x
													+ momentum_y * momentum_y + momentum_z * momentum_z;

		for (int i = 0; i < _NUMVECTORS; ++i)
		{
			double mom_dot_ei = (double)_CX_19[i] * momentum_x
									+ (double)_CY_19[i] * momentum_y
									+ (double)_CZ_19[i] * momentum_z;

			dev_fEq[i] = _EQMWEIGHTS_19[i]
							* (nn - (3.0 / 2.0) * momentumMagnitudeSquared * density_1
											+ (9.0 / 2.0) * density_1 * mom_dot_ei * mom_dot_ei + 3.0 * mom_dot_ei);
		}
		//-----------------------------------------------------------------------------------------------------------

		// d. Body Force case: Add details of any forcing scheme here - Evaluate force[i]
		// To do!!!
		//-----------------------------------------------------------------------------------------------------------

		// Collision step:
		// Single Relaxation Time approximation (LBGK)
		//double dev_fn[19];		// or maybe use the existing dev_ff[_NUMVECTORS] to minimise the memory requirements - Check and replace in the future

		// Evolution equation for the fi's here
		for (int i = 0; i < _NUMVECTORS; ++i)
		{
			//dev_fn[i] = dev_ff[i] + (dev_fEq[i] - dev_ff[i])/dev_tau; // + force[i];
			dev_ff[i] += (dev_ff[i] - dev_fEq[i]) * dev_minusInvTau; // Check if multiplying by dev_minusInvTau makes a difference
		}
		//__syncthreads(); // Check if needed!


		// --------------------------------------------------------------------------------
		// Streaming Step:
		// a. Load the streaming indices
		// b. The Iolet-Fluid links info
		// c. The ghost density
		// d. The inletNormal

		// a. Bulk Streaming indices: dev_NeighInd[19] here refers to the ACTUAL Streaming Array index (Data Address) in f_old and f_new
		//int64_t dev_NeighInd[19];

		// printf("Number of inlets: %d \n\n", nInlets);
		distribn_t ghost_dens; // = 0.0; //new distribn_t[nInlets];	// c. The ghost density
		float inletNormal_x, inletNormal_y, inletNormal_z;

	/*
		for(int LB_Dir=0; LB_Dir< _NUMVECTORS; LB_Dir++){

			// If we use the elements in GMem_int64_Neigh - then we access the memory address in fOld or fNew directly (not the fluid id)
			// (remember the memory layout in hemeLB is based on the site fluid index, i.e. f0[0], f1[0], f2[0], ..., fq[0] and for the Fluid Index Ind : f0[Ind], f1[Ind], f2[Ind], ..., fq[Ind]

			dev_NeighInd[LB_Dir] = GMem_int64_Neigh[(unsigned long long)LB_Dir * nArr_dbl + Ind]; // Read the streaming info here - Here Refers to Data Address NOT THE STREAMING FLUID ID!!!
		}
		__syncthreads();
	*/
		//
		// b. Iolet-Fluid links info:
		uint32_t Iolet_Intersect = GMem_uint32_Iolet_Link[Ind];


		// Read the ghost density and the inlet Normal
		// How do I distinguish which inlet ID do I have ??? Need to think about this... To do!!!
		// Need to pass this info based on the site Index (from the initialisation process. With given site ranges -> int boundaryId = site.GetIoletId();)

		// Access the info from the constant memory: _Iolets_Inlet_Inner[local_iolets_MaxSIZE], local_iolets_MaxSIZE = 6 cuda_params.h (Assume 2 max iolets per RANK)
		// Determine the IdInlet - Done!!!
		int IdInlet = INT32_MAX; // Iolet (Inlet/Outlet) ID
		if(num_local_Iolets==1){
			IdInlet = _Iolets_Outlet_Edge[0];// IdInlet = iolets_ID_range[0];
		}
		else{
			// Call a device function to determine which is the Iolet ID - using the iolets_ID_range Array
			// iolets_ID_range Array:
			//	a. Size: num_local_Iolets * 3
			// 	b. Iolet ID, Range of fluid IDs: [lower_limit, upper_limit)
			_determine_Iolet_ID(num_local_Iolets, _Iolets_Outlet_Edge, Ind, &IdInlet); // _determine_Iolet_ID(num_local_Iolets, iolets_ID_range, Ind, &IdInlet);
		}

		// Testing:
		if(IdInlet==INT32_MAX)
		{
			printf("Fluid_ID : %lld, ID_iolet: %d - Fluid NOT in IOLET range!!! \n\n", Ind, IdInlet);
		}
		/*else{
			printf("Fluid_ID : %lld, ID_iolet: %d \n\n", Ind, IdInlet);
		}
		*/

		//printf("Number of local Iolets = %d \n", num_local_Iolets);
/*
		for (int index = 0; index < num_local_Iolets; index++){
			printf(" Iolet ID: %d, lower_range: %lld, upper_range: %lld ", _Iolets_Inlet_Inner[3*index], _Iolets_Inlet_Inner[3*index+1], _Iolets_Inlet_Inner[3*index+2]);
		}
		printf("\n\n");
*/

		ghost_dens = GMem_ghostDensity[IdInlet];
		inletNormal_x = GMem_inletNormal[3*IdInlet];
		inletNormal_y = GMem_inletNormal[3*IdInlet+1];
		inletNormal_z = GMem_inletNormal[3*IdInlet+2];
	//		printf("ghost_dens[%d]: %.5f, inletNormal_x = %.5f, inletNormal_y = %.5f, inletNormal_z = %.5f  \n\n", IdInlet, ghost_dens, inletNormal_x, inletNormal_y, inletNormal_z);


		// Put the new populations after collision in the GMem_dbl array,
		// implementing the streaming step with Simple Bounce Back if Wall-Fluid link

		// fNew (dev_fn) populations:
		for (int LB_Dir = 0; LB_Dir < _NUMVECTORS; LB_Dir++)
		{
			unsigned mask = 1U << (LB_Dir - 1); // Needs to left shift the bits in mask so that I can then compare against the value in test_Wall_Intersect (To do: compare against test_bool_Wall_Intersect as well)
			bool is_Iolet_link = (Iolet_Intersect & mask);

			if(is_Iolet_link){	// ioletLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);

				//=============================================================================================================
				// Not valid in general! Need to change!!!
				// (IdInlet=0) Here we assume that we have only one inlet and the value of int boundaryId = site.GetIoletId() = 1. Need to change in the future!!!
				double component = velx*inletNormal_x + vely*inletNormal_y + velz*inletNormal_z;	// distribn_t component = (hydroVars.momentum / hydroVars.density).Dot(ioletNormal);

				// ghostHydrovars.momentum = ioletNormal * component * ghostDensity;
				double momentum_x = inletNormal_x * component * ghost_dens;
				double momentum_y = inletNormal_y * component * ghost_dens;
				double momentum_z = inletNormal_z * component * ghost_dens;


				//------------------------------------------------------------------------------------------------------
				// Calculate Feq[unstreamed_dir] - Only the direction that is necessary
				density_1 = 1.0 / ghost_dens;
				momentumMagnitudeSquared = momentum_x * momentum_x
													+ momentum_y * momentum_y + momentum_z * momentum_z;

				int unstreamed_dir = _InvDirections_19[LB_Dir];
				double mom_dot_ei = (double)_CX_19[unstreamed_dir] * momentum_x
									+ (double)_CY_19[unstreamed_dir] * momentum_y
									+ (double)_CZ_19[unstreamed_dir] * momentum_z;

				double dev_fEq_unstr = _EQMWEIGHTS_19[unstreamed_dir]
							* (ghost_dens - (3.0 / 2.0) * momentumMagnitudeSquared * density_1
											+ (9.0 / 2.0) * density_1 * mom_dot_ei * mom_dot_ei + 3.0 * mom_dot_ei);
				//------------------------------------------------------------------------------------------------------
				// Need to distinguish the int boundaryId = site.GetIoletId() correctly and pass the info (identify the proper ghost density and inlet-normals.
				//=============================================================================================================

				// printf("Site ID = %lld - Inlet in Dir: %d, Unstreamed direction: %d, fEq = %.5e \n\n", Ind, LB_Dir, unstreamed_dir, dev_fEq_unstr);

				// Case of NashZerothOrderPressure:
				// *latticeData->GetFNew(site.GetIndex() * LatticeType::NUMVECTORS + unstreamed) = ghostHydrovars.GetFEq()[unstreamed];
				GMem_dbl_fNew_b[(unsigned long long)unstreamed_dir * nArr_dbl + Ind] = dev_fEq_unstr;

			}
			else{ // bulkLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);

				// Use the Neighbouring Index given in GPUDataAddr_int64_Neigh_d, which is the actual streaming Array Index in f_new global memory
				int64_t dev_NeighInd = GMem_int64_Neigh[(unsigned long long)LB_Dir * nArr_dbl + Ind];

				// Save the post collision population in fNew
				GMem_dbl_fNew_b[dev_NeighInd] = dev_ff[LB_Dir];
				//
				// Debugging - Remove later
				// Check if it points to an address outside the (nFluid_nodes * _NUMVECTORS + 1+totalSharedFs )
				//if (dev_NeighInd[LB_Dir] >= (nArr_dbl*_NUMVECTORS+1+totalSharedFs)) printf("Error!!! Fluid Index = %lld, Stream.Dir.= %d, Max. Streaming addr = %lld Vs Stream. Addr.=%lld \n\n", Ind, LB_Dir, nArr_dbl*_NUMVECTORS+1+totalSharedFs, dev_NeighInd[LB_Dir] );
				//

				//---------------------------------------------------------------------------
			} // Closes the bulkLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);
		}
		//=============================================================================================


		// Write old density and velocity to memory -
		// Maybe use a different cuda kernel for these calculations (if saving the MacroVariables delays the collision/streaming kernel)
		// Check -  To do!!!
		if(time_Step%100==0){
			GMem_dbl_MacroVars[Ind] = nn;
			GMem_dbl_MacroVars[1ULL*nArr_dbl + Ind] = velx;
			GMem_dbl_MacroVars[2ULL*nArr_dbl + Ind] = vely;
			GMem_dbl_MacroVars[3ULL*nArr_dbl + Ind] = velz;
		}

	} // Ends the kernel GPU_Collide Type 4: mOutletCollision
	//==========================================================================================



	//**************************************************************
	// Kernel for the Collision step for the Lattice Boltzmann algorithm
	// 		Pressure BCs: Option NASHZEROTHORDERPRESSUREIOLET
	// 	Collision Types 3-4: mInletCollision & mOutletCollision: Inlet - Outlet BCs
	//												Pass a struct to the kernel containing the Iolet info
	//													struct Iolets Iolets_info
	//
	// Inlet BCs: specified with HEMELB_INLET_BOUNDARY in CMakeLists.txt
	//	Two Possible types of Inlet BCs:
	// 	1. NashZerothOrderPressure: Implement this first (see lb/streamers/NashZerothOrderPressureDelegate.h)
	//	2. LaddIolet: (see lb/streamers/LaddIoletDelegate.h)
	//
	// Implementation currently follows the memory arrangement of the data
	// 		by index LB, i.e. method (b)
	// Need to pass the information for the fluid-iolet links - Done!!!
	// 		This information is in ioletIntersection, see geometry/SiteDataBare.h
	//
	// This version uses the ACTUAL streaming address in global memory - NOT the fluid ID.
	//
	//**************************************************************
	__global__ void GPU_CollideStream_Iolets_NashZerothOrderPressure(distribn_t* GMem_dbl_fOld_b,
																distribn_t* GMem_dbl_fNew_b,
																distribn_t* GMem_dbl_MacroVars,
																int64_t* GMem_int64_Neigh,
																uint32_t* GMem_uint32_Iolet_Link,
																distribn_t* GMem_ghostDensity,
																float* GMem_inletNormal,
																int nInlets,
																uint64_t nArr_dbl,
																uint64_t lower_limit, uint64_t upper_limit, uint64_t totalSharedFs, int time_Step, int num_local_Iolets, Iolets Iolets_info)
	{
		unsigned long long Ind = blockIdx.x * blockDim.x + threadIdx.x;
		Ind = Ind + lower_limit;

		if(Ind >= upper_limit)
			return;

		// Load the distribution functions
		//f[19] and fEq[19]
		double dev_ff[19], dev_fEq[19];
		double nn = 0.0;	// density
		double momentum_x, momentum_y, momentum_z;
		momentum_x = momentum_y = momentum_z = 0.0;

		double velx, vely, velz;	// Fluid Velocity

		//-----------------------------------------------------------------------------------------------------------
		// 1. Read the fOld_GPU_b distr. functions
		// 2. Calculate the nessessary elements for calculating the equilibrium distribution functions
		// 		a. Calculate density
		// 		b. Calculate momentum - Needs to consider the case of body force as well - To do!!!
		for(int direction = 0; direction< _NUMVECTORS; direction++){
			dev_ff[direction] = GMem_dbl_fOld_b[(unsigned long long)direction * nArr_dbl + Ind];

			nn += dev_ff[direction];
			momentum_x += (double)_CX_19[direction] * dev_ff[direction];
			momentum_y += (double)_CY_19[direction] * dev_ff[direction];
			momentum_z += (double)_CZ_19[direction] * dev_ff[direction];
			//printf("Momentum: _x = %.5e, _y = %.5e, _z = %.5e \n\n", momentum_x, momentum_y, momentum_z);
		}


		// In the case of body force
		//momentum_x += 0.5 * _force_x;
		//momentum_y += 0.5 * _force_y;
		//momentum_z += 0.5 * _force_z;


		// Compute velocity components
		velx = momentum_x/nn;
		vely = momentum_y/nn;
		velz = momentum_z/nn;

		//-----------------------------------------------------------------------------------------------------------
		// c. Calculate equilibrium distr. functions
		double density_1 = 1.0 / nn;
		double momentumMagnitudeSquared = momentum_x * momentum_x
													+ momentum_y * momentum_y + momentum_z * momentum_z;

		for (int i = 0; i < _NUMVECTORS; ++i)
		{
			double mom_dot_ei = (double)_CX_19[i] * momentum_x
									+ (double)_CY_19[i] * momentum_y
									+ (double)_CZ_19[i] * momentum_z;

			dev_fEq[i] = _EQMWEIGHTS_19[i]
							* (nn - (3.0 / 2.0) * momentumMagnitudeSquared * density_1
											+ (9.0 / 2.0) * density_1 * mom_dot_ei * mom_dot_ei + 3.0 * mom_dot_ei);
		}
		//-----------------------------------------------------------------------------------------------------------

		// d. Body Force case: Add details of any forcing scheme here - Evaluate force[i]
		// To do!!!
		//-----------------------------------------------------------------------------------------------------------

		// Collision step:
		// Single Relaxation Time approximation (LBGK)
		//double dev_fn[19];		// or maybe use the existing dev_ff[_NUMVECTORS] to minimise the memory requirements - Check and replace in the future

		// Evolution equation for the fi's here
		for (int i = 0; i < _NUMVECTORS; ++i)
		{
			//dev_fn[i] = dev_ff[i] + (dev_fEq[i] - dev_ff[i])/dev_tau; // + force[i];
			dev_ff[i] += (dev_ff[i] - dev_fEq[i]) * dev_minusInvTau; // Check if multiplying by dev_minusInvTau makes a difference
		}
		//__syncthreads(); // Check if needed!


		// --------------------------------------------------------------------------------
		// Streaming Step:
		// a. Load the streaming indices
		// b. The Iolet-Fluid links info
		// c. The ghost density
		// d. The inletNormal

		// a. Bulk Streaming indices: dev_NeighInd[19] here refers to the ACTUAL Streaming Array index (Data Address) in f_old and f_new
		//int64_t dev_NeighInd[19];

		// printf("Number of inlets: %d \n\n", nInlets);
		distribn_t ghost_dens; // = 0.0; //new distribn_t[nInlets];	// c. The ghost density
		float inletNormal_x, inletNormal_y, inletNormal_z;

	/*
		for(int LB_Dir=0; LB_Dir< _NUMVECTORS; LB_Dir++){

			// If we use the elements in GMem_int64_Neigh - then we access the memory address in fOld or fNew directly (not the fluid id)
			// (remember the memory layout in hemeLB is based on the site fluid index, i.e. f0[0], f1[0], f2[0], ..., fq[0] and for the Fluid Index Ind : f0[Ind], f1[Ind], f2[Ind], ..., fq[Ind]

			dev_NeighInd[LB_Dir] = GMem_int64_Neigh[(unsigned long long)LB_Dir * nArr_dbl + Ind]; // Read the streaming info here - Here Refers to Data Address NOT THE STREAMING FLUID ID!!!
		}
		__syncthreads();
	*/
		//
		// b. Iolet-Fluid links info:
		uint32_t Iolet_Intersect = GMem_uint32_Iolet_Link[Ind];


		// Read the ghost density and the inlet Normal
		// How do I distinguish which inlet ID do I have ??? Need to think about this... To do!!!
		// Need to pass this info based on the site Index (from the initialisation process. With given site ranges -> int boundaryId = site.GetIoletId();)

		// Access the info from the constant memory: _Iolets_Inlet_Inner[local_iolets_MaxSIZE], local_iolets_MaxSIZE = 6 cuda_params.h (Assume 2 max iolets per RANK)
		// Determine the IdInlet - Done!!!
		int IdInlet = INT32_MAX; // Iolet (Inlet/Outlet) ID
		if(num_local_Iolets==1){
			IdInlet = Iolets_info.Iolets_ID_range[0];// IdInlet = iolets_ID_range[0];
		}
		else{
			// Call a device function to determine which is the Iolet ID - using the iolets_ID_range Array
			// iolets_ID_range Array:
			//	a. Size: num_local_Iolets * 3
			// 	b. Iolet ID, Range of fluid IDs: [lower_limit, upper_limit)
			_determine_Iolet_ID(num_local_Iolets, Iolets_info.Iolets_ID_range, Ind, &IdInlet); // _determine_Iolet_ID(num_local_Iolets, iolets_ID_range, Ind, &IdInlet);
		}

		// Testing:
		if(IdInlet==INT32_MAX)
		{
			printf("Fluid_ID : %lld, ID_iolet: %d - Fluid NOT in IOLET range!!! \n\n", Ind, IdInlet);
		}
		/*else{
			printf("Fluid_ID : %lld, ID_iolet: %d \n\n", Ind, IdInlet);
		}
		*/

		//printf("Number of local Iolets = %d \n", num_local_Iolets);
/*
		for (int index = 0; index < num_local_Iolets; index++){
			printf(" Iolet ID: %d, lower_range: %lld, upper_range: %lld ", _Iolets_Inlet_Inner[3*index], _Iolets_Inlet_Inner[3*index+1], _Iolets_Inlet_Inner[3*index+2]);
		}
		printf("\n\n");
*/

		ghost_dens = GMem_ghostDensity[IdInlet];
		inletNormal_x = GMem_inletNormal[3*IdInlet];
		inletNormal_y = GMem_inletNormal[3*IdInlet+1];
		inletNormal_z = GMem_inletNormal[3*IdInlet+2];
	//		printf("ghost_dens[%d]: %.5f, inletNormal_x = %.5f, inletNormal_y = %.5f, inletNormal_z = %.5f  \n\n", IdInlet, ghost_dens, inletNormal_x, inletNormal_y, inletNormal_z);


		// Put the new populations after collision in the GMem_dbl array,
		// implementing the streaming step with Simple Bounce Back if Wall-Fluid link

		// fNew (dev_fn) populations:
		for (int LB_Dir = 0; LB_Dir < _NUMVECTORS; LB_Dir++)
		{
			unsigned mask = 1U << (LB_Dir - 1); // Needs to left shift the bits in mask so that I can then compare against the value in test_Wall_Intersect (To do: compare against test_bool_Wall_Intersect as well)
			bool is_Iolet_link = (Iolet_Intersect & mask);

			if(is_Iolet_link){	// ioletLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);

				//=============================================================================================================
				// Not valid in general! Need to change!!!
				// (IdInlet=0) Here we assume that we have only one inlet and the value of int boundaryId = site.GetIoletId() = 1. Need to change in the future!!!
				double component = velx*inletNormal_x + vely*inletNormal_y + velz*inletNormal_z;	// distribn_t component = (hydroVars.momentum / hydroVars.density).Dot(ioletNormal);

				// ghostHydrovars.momentum = ioletNormal * component * ghostDensity;
				double momentum_x = inletNormal_x * component * ghost_dens;
				double momentum_y = inletNormal_y * component * ghost_dens;
				double momentum_z = inletNormal_z * component * ghost_dens;


				//------------------------------------------------------------------------------------------------------
				// Calculate Feq[unstreamed_dir] - Only the direction that is necessary
				density_1 = 1.0 / ghost_dens;
				momentumMagnitudeSquared = momentum_x * momentum_x
													+ momentum_y * momentum_y + momentum_z * momentum_z;

				int unstreamed_dir = _InvDirections_19[LB_Dir];
				double mom_dot_ei = (double)_CX_19[unstreamed_dir] * momentum_x
									+ (double)_CY_19[unstreamed_dir] * momentum_y
									+ (double)_CZ_19[unstreamed_dir] * momentum_z;

				double dev_fEq_unstr = _EQMWEIGHTS_19[unstreamed_dir]
							* (ghost_dens - (3.0 / 2.0) * momentumMagnitudeSquared * density_1
											+ (9.0 / 2.0) * density_1 * mom_dot_ei * mom_dot_ei + 3.0 * mom_dot_ei);
				//------------------------------------------------------------------------------------------------------
				// Need to distinguish the int boundaryId = site.GetIoletId() correctly and pass the info (identify the proper ghost density and inlet-normals.
				//=============================================================================================================

				// printf("Site ID = %lld - Inlet in Dir: %d, Unstreamed direction: %d, fEq = %.5e \n\n", Ind, LB_Dir, unstreamed_dir, dev_fEq_unstr);

				// Case of NashZerothOrderPressure:
				// *latticeData->GetFNew(site.GetIndex() * LatticeType::NUMVECTORS + unstreamed) = ghostHydrovars.GetFEq()[unstreamed];
				GMem_dbl_fNew_b[(unsigned long long)unstreamed_dir * nArr_dbl + Ind] = dev_fEq_unstr;

			}
			else{ // bulkLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);

				// Use the Neighbouring Index given in GPUDataAddr_int64_Neigh_d, which is the actual streaming Array Index in f_new global memory
				int64_t dev_NeighInd = GMem_int64_Neigh[(unsigned long long)LB_Dir * nArr_dbl + Ind];

				// Save the post collision population in fNew
				GMem_dbl_fNew_b[dev_NeighInd] = dev_ff[LB_Dir];
				//
				// Debugging - Remove later
				// Check if it points to an address outside the (nFluid_nodes * _NUMVECTORS + 1+totalSharedFs )
				//if (dev_NeighInd[LB_Dir] >= (nArr_dbl*_NUMVECTORS+1+totalSharedFs)) printf("Error!!! Fluid Index = %lld, Stream.Dir.= %d, Max. Streaming addr = %lld Vs Stream. Addr.=%lld \n\n", Ind, LB_Dir, nArr_dbl*_NUMVECTORS+1+totalSharedFs, dev_NeighInd[LB_Dir] );
				//

				//---------------------------------------------------------------------------
			} // Closes the bulkLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);
		}
		//=============================================================================================


		// Write old density and velocity to memory -
		// Maybe use a different cuda kernel for these calculations (if saving the MacroVariables delays the collision/streaming kernel)
		// Check -  To do!!!
		if(time_Step%_Send_MacroVars_DtH==0){
			GMem_dbl_MacroVars[Ind] = nn;
			GMem_dbl_MacroVars[1ULL*nArr_dbl + Ind] = velx;
			GMem_dbl_MacroVars[2ULL*nArr_dbl + Ind] = vely;
			GMem_dbl_MacroVars[3ULL*nArr_dbl + Ind] = velz;
		}

	} // Ends the kernel GPU_Collide Type 4: mOutletCollision
	//==========================================================================================

//********************************************************************************************
// Currently in use:

//**************************************************************
// Kernel for the Collision step for the Lattice Boltzmann algorithm
// 		Pressure BCs: Option NASHZEROTHORDERPRESSUREIOLET
// 	Collision Types 3-4: mInletCollision & mOutletCollision: Inlet - Outlet BCs
//												Iolet's info accessed from GPU global memory
//
// Inlet BCs: specified with HEMELB_INLET_BOUNDARY in CMakeLists.txt
//	Two Possible types of Inlet BCs:
// 	1. NashZerothOrderPressure: Implement this first (see lb/streamers/NashZerothOrderPressureDelegate.h)
//	2. LaddIolet: (see lb/streamers/LaddIoletDelegate.h)
//
// Implementation currently follows the memory arrangement of the data
// 		by index LB, i.e. method (b)
// Need to pass the information for the fluid-iolet links - Done!!!
// 		This information is in ioletIntersection, see geometry/SiteDataBare.h
//
// This version uses the ACTUAL streaming address in global memory - NOT the fluid ID.
//
//**************************************************************
__global__ void GPU_CollideStream_Iolets_NashZerothOrderPressure_v2(distribn_t* GMem_dbl_fOld_b,
															distribn_t* GMem_dbl_fNew_b,
															distribn_t* GMem_dbl_MacroVars,
															int64_t* GMem_int64_Neigh,
															uint32_t* GMem_uint32_Iolet_Link,
															distribn_t* GMem_ghostDensity,
															float* GMem_inletNormal,
															int nInlets,
															uint64_t nArr_dbl,
															uint64_t lower_limit, uint64_t upper_limit, uint64_t totalSharedFs, int time_Step, int num_local_Iolets, site_t* GMem_Iolets_info)
{
	unsigned long long Ind = blockIdx.x * blockDim.x + threadIdx.x;
	Ind = Ind + lower_limit;

	if(Ind >= upper_limit)
		return;

	// Load the distribution functions
	//f[19] and fEq[19]
	double dev_ff[19], dev_fEq[19];
	double nn = 0.0;	// density
	double momentum_x, momentum_y, momentum_z;
	momentum_x = momentum_y = momentum_z = 0.0;

	double velx, vely, velz;	// Fluid Velocity

	//-----------------------------------------------------------------------------------------------------------
	// 1. Read the fOld_GPU_b distr. functions
	// 2. Calculate the nessessary elements for calculating the equilibrium distribution functions
	// 		a. Calculate density
	// 		b. Calculate momentum - Needs to consider the case of body force as well - To do!!!
	for(int direction = 0; direction< _NUMVECTORS; direction++){
		dev_ff[direction] = GMem_dbl_fOld_b[(unsigned long long)direction * nArr_dbl + Ind];

		nn += dev_ff[direction];
		momentum_x += (double)_CX_19[direction] * dev_ff[direction];
		momentum_y += (double)_CY_19[direction] * dev_ff[direction];
		momentum_z += (double)_CZ_19[direction] * dev_ff[direction];
		//printf("Momentum: _x = %.5e, _y = %.5e, _z = %.5e \n\n", momentum_x, momentum_y, momentum_z);
	}


	// In the case of body force
	//momentum_x += 0.5 * _force_x;
	//momentum_y += 0.5 * _force_y;
	//momentum_z += 0.5 * _force_z;


	// Compute velocity components
	velx = momentum_x/nn;
	vely = momentum_y/nn;
	velz = momentum_z/nn;

	//-----------------------------------------------------------------------------------------------------------
	// c. Calculate equilibrium distr. functions
	double density_1 = 1.0 / nn;
	double momentumMagnitudeSquared = momentum_x * momentum_x
												+ momentum_y * momentum_y + momentum_z * momentum_z;

	for (int i = 0; i < _NUMVECTORS; ++i)
	{
		double mom_dot_ei = (double)_CX_19[i] * momentum_x
								+ (double)_CY_19[i] * momentum_y
								+ (double)_CZ_19[i] * momentum_z;

		dev_fEq[i] = _EQMWEIGHTS_19[i]
						* (nn - (3.0 / 2.0) * momentumMagnitudeSquared * density_1
										+ (9.0 / 2.0) * density_1 * mom_dot_ei * mom_dot_ei + 3.0 * mom_dot_ei);
	}
	//-----------------------------------------------------------------------------------------------------------

	// d. Body Force case: Add details of any forcing scheme here - Evaluate force[i]
	// To do!!!
	//-----------------------------------------------------------------------------------------------------------

	// Collision step:
	// Single Relaxation Time approximation (LBGK)
	//double dev_fn[19];		// or maybe use the existing dev_ff[_NUMVECTORS] to minimise the memory requirements - Check and replace in the future

	// Evolution equation for the fi's here
	for (int i = 0; i < _NUMVECTORS; ++i)
	{
		//dev_fn[i] = dev_ff[i] + (dev_fEq[i] - dev_ff[i])/dev_tau; // + force[i];
		dev_ff[i] += (dev_ff[i] - dev_fEq[i]) * dev_minusInvTau; // Check if multiplying by dev_minusInvTau makes a difference
	}
	//__syncthreads(); // Check if needed!


	// --------------------------------------------------------------------------------
	// Streaming Step:
	// a. Load the streaming indices
	// b. The Iolet-Fluid links info
	// c. The ghost density
	// d. The inletNormal

	// a. Bulk Streaming indices: dev_NeighInd[19] here refers to the ACTUAL Streaming Array index (Data Address) in f_old and f_new
	//int64_t dev_NeighInd[19];

	// printf("Number of inlets: %d \n\n", nInlets);
	distribn_t ghost_dens; // = 0.0; //new distribn_t[nInlets];	// c. The ghost density
	float inletNormal_x, inletNormal_y, inletNormal_z;

/*
	for(int LB_Dir=0; LB_Dir< _NUMVECTORS; LB_Dir++){

		// If we use the elements in GMem_int64_Neigh - then we access the memory address in fOld or fNew directly (not the fluid id)
		// (remember the memory layout in hemeLB is based on the site fluid index, i.e. f0[0], f1[0], f2[0], ..., fq[0] and for the Fluid Index Ind : f0[Ind], f1[Ind], f2[Ind], ..., fq[Ind]

		dev_NeighInd[LB_Dir] = GMem_int64_Neigh[(unsigned long long)LB_Dir * nArr_dbl + Ind]; // Read the streaming info here - Here Refers to Data Address NOT THE STREAMING FLUID ID!!!
	}
	__syncthreads();
*/
	//
	// b. Iolet-Fluid links info:
	uint32_t Iolet_Intersect = GMem_uint32_Iolet_Link[Ind];

	// Read the Iolet info (iolet ids and fluid sites range) from GMem_Iolets_info
	//	extern __shared__ int s[];
	site_t *Iolet_info = new site_t[3*num_local_Iolets];
	for (int index = 0; index< (3*num_local_Iolets); index++)
	{
		Iolet_info[index] = GMem_Iolets_info[index];
	}

	// Read the ghost density and the inlet Normal
	// How do I distinguish which inlet ID do I have ??? Need to think about this... To do!!!
	// Need to pass this info based on the site Index (from the initialisation process. With given site ranges -> int boundaryId = site.GetIoletId();)

	// Determine the IdInlet - Done!!!
	int IdInlet = INT32_MAX; // Iolet (Inlet/Outlet) ID
	if(num_local_Iolets==1){
		IdInlet = Iolet_info[0]; //Iolets_info.Iolets_ID_range[0];// IdInlet = iolets_ID_range[0];
	}
	else{
		// Call a device function to determine which is the Iolet ID - using the iolets_ID_range Array
		// iolets_ID_range Array:
		//	a. Size: num_local_Iolets * 3
		// 	b. Iolet ID, Range of fluid IDs: [lower_limit, upper_limit)
		// TODO: Replace this: _determine_Iolet_ID(num_local_Iolets, Iolets_info.Iolets_ID_range, Ind, &IdInlet); // _determine_Iolet_ID(num_local_Iolets, iolets_ID_range, Ind, &IdInlet);
		_determine_Iolet_ID(num_local_Iolets, Iolet_info, Ind, &IdInlet);
	}

	// Testing:
	if(IdInlet==INT32_MAX)
	{
		printf("Fluid_ID : %lld, ID_iolet: %d - Fluid NOT in IOLET range!!! \n\n", Ind, IdInlet);
	}
	/*else{
		printf("Fluid_ID : %lld, ID_iolet: %d \n\n", Ind, IdInlet);
	}
	*/

	//printf("Number of local Iolets = %d \n", num_local_Iolets);
/*
	for (int index = 0; index < num_local_Iolets; index++){
		printf(" Iolet ID: %d, lower_range: %lld, upper_range: %lld ", _Iolets_Inlet_Inner[3*index], _Iolets_Inlet_Inner[3*index+1], _Iolets_Inlet_Inner[3*index+2]);
	}
	printf("\n\n");
*/

	ghost_dens = GMem_ghostDensity[IdInlet];
	inletNormal_x = GMem_inletNormal[3*IdInlet];
	inletNormal_y = GMem_inletNormal[3*IdInlet+1];
	inletNormal_z = GMem_inletNormal[3*IdInlet+2];
//		printf("ghost_dens[%d]: %.5f, inletNormal_x = %.5f, inletNormal_y = %.5f, inletNormal_z = %.5f  \n\n", IdInlet, ghost_dens, inletNormal_x, inletNormal_y, inletNormal_z);


	// Put the new populations after collision in the GMem_dbl array,
	// implementing the streaming step with Simple Bounce Back if Wall-Fluid link

	// fNew (dev_fn) populations:
	for (int LB_Dir = 0; LB_Dir < _NUMVECTORS; LB_Dir++)
	{
		unsigned mask = 1U << (LB_Dir - 1); // Needs to left shift the bits in mask so that I can then compare against the value in test_Wall_Intersect (To do: compare against test_bool_Wall_Intersect as well)
		bool is_Iolet_link = (Iolet_Intersect & mask);

		if(is_Iolet_link){	// ioletLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);

			//=============================================================================================================
			// Not valid in general! Need to change!!!
			// (IdInlet=0) Here we assume that we have only one inlet and the value of int boundaryId = site.GetIoletId() = 1. Need to change in the future!!!
			double component = velx*inletNormal_x + vely*inletNormal_y + velz*inletNormal_z;	// distribn_t component = (hydroVars.momentum / hydroVars.density).Dot(ioletNormal);

			// ghostHydrovars.momentum = ioletNormal * component * ghostDensity;
			double momentum_x = inletNormal_x * component * ghost_dens;
			double momentum_y = inletNormal_y * component * ghost_dens;
			double momentum_z = inletNormal_z * component * ghost_dens;


			//------------------------------------------------------------------------------------------------------
			// Calculate Feq[unstreamed_dir] - Only the direction that is necessary
			density_1 = 1.0 / ghost_dens;
			momentumMagnitudeSquared = momentum_x * momentum_x
												+ momentum_y * momentum_y + momentum_z * momentum_z;

			int unstreamed_dir = _InvDirections_19[LB_Dir];
			double mom_dot_ei = (double)_CX_19[unstreamed_dir] * momentum_x
								+ (double)_CY_19[unstreamed_dir] * momentum_y
								+ (double)_CZ_19[unstreamed_dir] * momentum_z;

			double dev_fEq_unstr = _EQMWEIGHTS_19[unstreamed_dir]
						* (ghost_dens - (3.0 / 2.0) * momentumMagnitudeSquared * density_1
										+ (9.0 / 2.0) * density_1 * mom_dot_ei * mom_dot_ei + 3.0 * mom_dot_ei);
			//------------------------------------------------------------------------------------------------------
			// Need to distinguish the int boundaryId = site.GetIoletId() correctly and pass the info (identify the proper ghost density and inlet-normals.
			//=============================================================================================================

			// printf("Site ID = %lld - Inlet in Dir: %d, Unstreamed direction: %d, fEq = %.5e \n\n", Ind, LB_Dir, unstreamed_dir, dev_fEq_unstr);

			// Case of NashZerothOrderPressure:
			// *latticeData->GetFNew(site.GetIndex() * LatticeType::NUMVECTORS + unstreamed) = ghostHydrovars.GetFEq()[unstreamed];
			GMem_dbl_fNew_b[(unsigned long long)unstreamed_dir * nArr_dbl + Ind] = dev_fEq_unstr;

		}
		else{ // bulkLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);

			// Use the Neighbouring Index given in GPUDataAddr_int64_Neigh_d, which is the actual streaming Array Index in f_new global memory
			int64_t dev_NeighInd = GMem_int64_Neigh[(unsigned long long)LB_Dir * nArr_dbl + Ind];

			// Save the post collision population in fNew
			GMem_dbl_fNew_b[dev_NeighInd] = dev_ff[LB_Dir];
			//
			// Debugging - Remove later
			// Check if it points to an address outside the (nFluid_nodes * _NUMVECTORS + 1+totalSharedFs )
			//if (dev_NeighInd[LB_Dir] >= (nArr_dbl*_NUMVECTORS+1+totalSharedFs)) printf("Error!!! Fluid Index = %lld, Stream.Dir.= %d, Max. Streaming addr = %lld Vs Stream. Addr.=%lld \n\n", Ind, LB_Dir, nArr_dbl*_NUMVECTORS+1+totalSharedFs, dev_NeighInd[LB_Dir] );
			//

			//---------------------------------------------------------------------------
		} // Closes the bulkLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);
	}
	//=============================================================================================


	// Write old density and velocity to memory -
	// Maybe use a different cuda kernel for these calculations (if saving the MacroVariables delays the collision/streaming kernel)
	// Check -  To do!!!
	if(time_Step%_Send_MacroVars_DtH==0){
		GMem_dbl_MacroVars[Ind] = nn;
		GMem_dbl_MacroVars[1ULL*nArr_dbl + Ind] = velx;
		GMem_dbl_MacroVars[2ULL*nArr_dbl + Ind] = vely;
		GMem_dbl_MacroVars[3ULL*nArr_dbl + Ind] = velz;
	}

	delete[] Iolet_info;
} // Ends the kernel GPU_Collide Type 4: mOutletCollision
//==========================================================================================




	//**************************************************************
	// Kernel for the Collision step for the Lattice Boltzmann algorithm
	// 		Velocity BCs: Option LADDIOLET
	// 	Collision Types 3-4: mInletCollision & mOutletCollision: Inlet - Outlet BCs
	//
	// Inlet BCs: specified with HEMELB_INLET_BOUNDARY in CMakeLists.txt
	//	Two Possible types of Inlet BCs:
	// 	1. NashZerothOrderPressure: Implement this first (see lb/streamers/NashZerothOrderPressureDelegate.h)
	//	2. LaddIolet: (see lb/streamers/LaddIoletDelegate.h)
	//
	// Implementation currently follows the memory arrangement of the data
	// by index LB, i.e. method (b)
	// Need to pass the information for the fluid-iolet links - Done!!!
	// This information is in ioletIntersection, see geometry/SiteDataBare.h
	//
	// This version uses the ACTUAL streaming address in global memory - NOT the fluid ID.

	// Note regarding the wall mom:
	// TODO: Pass the boolean variable: CollisionType::CKernel::LatticeType::IsLatticeCompressible()
	// Remember that the wall mom. does not include the correction (multiplication by local density) If Compressible:
	//**************************************************************
	__global__ void GPU_CollideStream_Iolets_Ladd_VelBCs(distribn_t* GMem_dbl_fOld_b,
																distribn_t* GMem_dbl_fNew_b,
																distribn_t* GMem_dbl_MacroVars,
																int64_t* GMem_int64_Neigh,
																uint32_t* GMem_uint32_Iolet_Link,
																uint64_t nArr_dbl,
																distribn_t* GMem_dbl_WallMom, uint64_t nArr_wallMom,
																uint64_t lower_limit, uint64_t upper_limit, uint64_t totalSharedFs, int time_Step)
	{
		unsigned long long Ind = blockIdx.x * blockDim.x + threadIdx.x;
		Ind = Ind + lower_limit;

		if(Ind >= upper_limit)
			return;

		// Load the distribution functions
		//f[19] and fEq[19]
		double dev_ff[19], dev_fEq[19];
		double nn = 0.0;	// density
		double momentum_x, momentum_y, momentum_z;
		momentum_x = momentum_y = momentum_z = 0.0;

		double velx, vely, velz;	// Fluid Velocity

		//-----------------------------------------------------------------------------------------------------------
		// 1. Read the fOld_GPU_b distr. functions
		// 2. Calculate the nessessary elements for calculating the equilibrium distribution functions
		// 		a. Calculate density
		// 		b. Calculate momentum - Needs to consider the case of body force as well - To do!!!
		for(int direction = 0; direction< _NUMVECTORS; direction++){
			dev_ff[direction] = GMem_dbl_fOld_b[(unsigned long long)direction * nArr_dbl + Ind];

			nn += dev_ff[direction];
			momentum_x += (double)_CX_19[direction] * dev_ff[direction];
			momentum_y += (double)_CY_19[direction] * dev_ff[direction];
			momentum_z += (double)_CZ_19[direction] * dev_ff[direction];
			//printf("Momentum: _x = %.5e, _y = %.5e, _z = %.5e \n\n", momentum_x, momentum_y, momentum_z);
		}

		// In the case of body force
		//momentum_x += 0.5 * _force_x;
		//momentum_y += 0.5 * _force_y;
		//momentum_z += 0.5 * _force_z;

		// Compute velocity components
		velx = momentum_x/nn;
		vely = momentum_y/nn;
		velz = momentum_z/nn;

		//-----------------------------------------------------------------------------------------------------------
		// c. Calculate equilibrium distr. functions
		double density_1 = 1.0 / nn;
		double momentumMagnitudeSquared = momentum_x * momentum_x
													+ momentum_y * momentum_y + momentum_z * momentum_z;

		for (int i = 0; i < _NUMVECTORS; ++i)
		{
			double mom_dot_ei = (double)_CX_19[i] * momentum_x
									+ (double)_CY_19[i] * momentum_y
									+ (double)_CZ_19[i] * momentum_z;

			dev_fEq[i] = _EQMWEIGHTS_19[i]
							* (nn - (3.0 / 2.0) * momentumMagnitudeSquared * density_1
											+ (9.0 / 2.0) * density_1 * mom_dot_ei * mom_dot_ei + 3.0 * mom_dot_ei);
		}
		//-----------------------------------------------------------------------------------------------------------

		// d. Body Force case: Add details of any forcing scheme here - Evaluate force[i]
		// To do!!!
		//-----------------------------------------------------------------------------------------------------------

		// Collision step:
		// Single Relaxation Time approximation (LBGK)
		//double dev_fn[19];		// or maybe use the existing dev_ff[_NUMVECTORS] to minimise the memory requirements - Check and replace in the future

		// Evolution equation for the fi's here
		for (int i = 0; i < _NUMVECTORS; ++i)
		{
			//dev_fn[i] = dev_ff[i] + (dev_fEq[i] - dev_ff[i])/dev_tau; // + force[i];
			dev_ff[i] += (dev_ff[i] - dev_fEq[i]) * dev_minusInvTau; // Check if multiplying by dev_minusInvTau makes a difference
		}
		//__syncthreads(); // Check if needed!


		// --------------------------------------------------------------------------------
		// Streaming Step:
		// 	a. Load the streaming indices
		// 	b. The Iolet-Fluid links info
		/**
				If the link is an Iolet-Fluid link
		 		c. Load the wallMom array
		 		d. Compute the correction to the bounced back part of the distr. functions:
				Hence, need to have the following:
					d.1. LatticeType::EQMWEIGHTS[LB_dir]
					d.2. LatticeType::CX[LB_dir], LatticeType::CY[LB_dir], LatticeType::CZ[LB_dir],
					d.3. Cs2
					d.4. Bounced-back index: just the INVERSEDIRECTIONS is sufficient
		*/

		//
		// b. Iolet-Fluid links info:
		uint32_t Iolet_Intersect = GMem_uint32_Iolet_Link[Ind];

		// Put the new populations after collision in the GMem_dbl array,
		// implementing the streaming step with Simple Bounce Back if Wall-Fluid link

		// fNew (dev_fn) populations:
		for (int LB_Dir = 0; LB_Dir < _NUMVECTORS; LB_Dir++)
		{
			unsigned mask = 1U << (LB_Dir - 1); // Needs to left shift the bits in mask so that I can then compare against the value in test_Wall_Intersect (To do: compare against test_bool_Wall_Intersect as well)
			bool is_Iolet_link = (Iolet_Intersect & mask);

			if(is_Iolet_link){	// ioletLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);
				//=============================================================================================================
				// c. Load the WallMom info - Note: We follow Method b for the data layout
				site_t siteCount = upper_limit-lower_limit;
				//site_t nArr_wallMom = siteCount * (_NUMVECTORS-1); // Number of elements of type distribn_t(double)

				distribn_t WallMom_x, WallMom_y, WallMom_z;
				site_t shifted_Fluid_Ind = Ind - lower_limit;
				WallMom_x = GMem_dbl_WallMom[(unsigned long long)(LB_Dir - 1) * siteCount + shifted_Fluid_Ind];
				WallMom_y = GMem_dbl_WallMom[1ULL*nArr_wallMom + (unsigned long long)(LB_Dir - 1) * siteCount + shifted_Fluid_Ind];
				WallMom_z = GMem_dbl_WallMom[2ULL*nArr_wallMom + (unsigned long long)(LB_Dir - 1) * siteCount + shifted_Fluid_Ind];

				//-----------------------
				// TODO: Pass the boolean variable: CollisionType::CKernel::LatticeType::IsLatticeCompressible()
				// Remember that the wall mom. does not include the correction (multiplication by local density) If Compressible:
				WallMom_x *= nn;
				WallMom_y *= nn;
				WallMom_z *= nn;
				//-----------------------

				distribn_t correction = 2. * _EQMWEIGHTS_19[LB_Dir]
				                * (WallMom_x * _CX_19[LB_Dir] + WallMom_y * _CY_19[LB_Dir] + WallMom_z * _CZ_19[LB_Dir]) / _Cs2;

				int unstreamed_dir = _InvDirections_19[LB_Dir];

				GMem_dbl_fNew_b[(unsigned long long)unstreamed_dir * nArr_dbl + Ind] = dev_ff[LB_Dir] - correction;

				/*
				// Implement the following:
				// 	Iolet is in the LB direction = LB_dir
				* (latticeData->GetFNew(SimpleBounceBackDelegate<CollisionImpl>::GetBBIndex(site.GetIndex(), LB_dir))) =
				                hydroVars.GetFPostCollision()[LB_dir] - correction;
				*/
				//=============================================================================================================
			}
			else{ // bulkLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);

				// Use the Neighbouring Index given in GPUDataAddr_int64_Neigh_d, which is the actual streaming Array Index in f_new global memory
				int64_t dev_NeighInd = GMem_int64_Neigh[(unsigned long long)LB_Dir * nArr_dbl + Ind];

				// Save the post collision population in fNew
				GMem_dbl_fNew_b[dev_NeighInd] = dev_ff[LB_Dir];
				//
				// Debugging - Remove later
				// Check if it points to an address outside the (nFluid_nodes * _NUMVECTORS + 1+totalSharedFs )
				//if (dev_NeighInd[LB_Dir] >= (nArr_dbl*_NUMVECTORS+1+totalSharedFs)) printf("Error!!! Fluid Index = %lld, Stream.Dir.= %d, Max. Streaming addr = %lld Vs Stream. Addr.=%lld \n\n", Ind, LB_Dir, nArr_dbl*_NUMVECTORS+1+totalSharedFs, dev_NeighInd[LB_Dir] );
				//

				//---------------------------------------------------------------------------
			} // Closes the bulkLinkDelegate.StreamLink(lbmParams, latDat, site, hydroVars, ii);
		}
		//=============================================================================================

		// Write old density and velocity to memory -
		// Maybe use a different cuda kernel for these calculations (if saving the MacroVariables delays the collision/streaming kernel)
		// Check -  To do!!!
		if(time_Step%_Send_MacroVars_DtH==0){
			GMem_dbl_MacroVars[Ind] = nn;
			GMem_dbl_MacroVars[1ULL*nArr_dbl + Ind] = velx;
			GMem_dbl_MacroVars[2ULL*nArr_dbl + Ind] = vely;
			GMem_dbl_MacroVars[3ULL*nArr_dbl + Ind] = velz;
		}

	} // Ends the kernel GPU_Collide Type 3-4: mInletCollision - mOutletCollision & Velocity BCs
	//==========================================================================================



#endif // #ifdef HEMELB_USE_GPU
} // namespace hemelb
