#include "hip/hip_runtime.h"
// This file is part of HemeLB and is Copyright (C)
// the HemeLB team and/or their institutions, as detailed in the
// file AUTHORS. This software is provided under the terms of the
// license in the file LICENSE.

#include "SimulationMaster.h"
#include "configuration/SimConfig.h"
#include "extraction/PropertyActor.h"
#include "extraction/LbDataSourceIterator.h"
#include "io/writers/xdr/XdrFileWriter.h"
#include "util/utilityFunctions.h"
#include "geometry/GeometryReader.h"
#include "geometry/LatticeData.h"
#include "util/fileutils.h"
#include "log/Logger.h"
#include "lb/HFunction.h"
#include "io/xml/XmlAbstractionLayer.h"
#include "colloids/ColloidController.h"
#include "net/BuildInfo.h"
#include "net/IOCommunicator.h"
#include "colloids/BodyForces.h"
#include "colloids/BoundaryConditions.h"

#include "net/MpiCommunicator.h"


#ifdef HEMELB_USE_GPU
//#include "cuda_kernels_def_decl/initialise_GPU.h"
#endif

#include <map>
#include <limits>
#include <cstdlib>

/**
 * Constructor for the SimulationMaster class
 *
 * Initialises member variables including the network topology
 * object.
 */

double hemelb::tau_;

SimulationMaster::SimulationMaster(hemelb::configuration::CommandLine & options, const hemelb::net::IOCommunicator& ioComm) :
	ioComms(ioComm), timings(ioComm), build_info(), communicationNet(ioComm) {
	timings[hemelb::reporting::Timers::total].Start();

	latticeData = NULL;

	colloidController = NULL;
	latticeBoltzmannModel = NULL;
	propertyDataSource = NULL;
	propertyExtractor = NULL;
	simulationState = NULL;
	stepManager = NULL;
	netConcern = NULL;
	neighbouringDataManager = NULL;
	imagesPerSimulation = options.NumberOfImages();

	fileManager = new hemelb::io::PathManager(options, IsCurrentProcTheIOProc(), GetProcessorCount());
	simConfig = hemelb::configuration::SimConfig::New(fileManager->GetInputFile());
	unitConverter = &simConfig->GetUnitConverter();
	monitoringConfig = simConfig->GetMonitoringConfiguration();

	fileManager->SaveConfiguration(simConfig);

	Initialise();


	if (IsCurrentProcTheIOProc()) {
		reporter = new hemelb::reporting::Reporter(
			fileManager->GetReportPath(),
			fileManager->GetInputFile());
		reporter->AddReportable(&build_info);

		if (monitoringConfig->doIncompressibilityCheck) {
			reporter->AddReportable(incompressibilityChecker);
		}

		reporter->AddReportable(&timings);
		reporter->AddReportable(latticeData);
		reporter->AddReportable(simulationState);
	}
}

/**
 * Destructor for the SimulationMaster class.
 *
 * Deallocates dynamically allocated memory to contained classes.
 */
SimulationMaster::~SimulationMaster() {

	delete latticeData;
	delete colloidController;
	delete latticeBoltzmannModel;
	delete inletValues;
	delete outletValues;
	delete propertyExtractor;
	delete propertyDataSource;
	delete stabilityTester;
	delete entropyTester;
	delete simulationState;
	delete incompressibilityChecker;
	delete neighbouringDataManager;

	delete simConfig;
	delete fileManager;

	if (IsCurrentProcTheIOProc()) {
		delete reporter;
	}

	delete stepManager;
	delete netConcern;
}

/**
 * Returns true if the current processor is the dedicated I/O
 * processor.
 */
bool SimulationMaster::IsCurrentProcTheIOProc() {
	return ioComms.OnIORank();
}

/**
 * Returns the number of processors involved in the simulation.
 */
int SimulationMaster::GetProcessorCount() {
	return ioComms.Size();
}

/**
 * Initialises various elements of the simulation
 */
void SimulationMaster::Initialise() {

	simulationState = new hemelb::lb::SimulationState(
		simConfig->GetTimeStepLength(),
		simConfig->GetTotalTimeSteps());

	timings[hemelb::reporting::Timers::latDatInitialise].Start();

	// Use a reader to read in the file.
	hemelb::log::Logger::Log<hemelb::log::Info, hemelb::log::Singleton>("INITIALISE");
	hemelb::log::Logger::Log<hemelb::log::Info, hemelb::log::Singleton>("----------");
	hemelb::log::Logger::Log<hemelb::log::Info, hemelb::log::Singleton>("--> loading input and decomposing geometry");
	hemelb::geometry::GeometryReader reader(
		latticeType::GetLatticeInfo(),
		timings, ioComms);
	hemelb::geometry::Geometry readGeometryData =
		reader.LoadAndDecompose(simConfig->GetDataFilePath());

	// Create a new lattice based on that info and return it.
	hemelb::log::Logger::Log<hemelb::log::Info, hemelb::log::Singleton>("--> lattice data");
	latticeData = new hemelb::geometry::LatticeData(latticeType::GetLatticeInfo(),
			readGeometryData,
			ioComms);

	timings[hemelb::reporting::Timers::latDatInitialise].Stop();

	hemelb::log::Logger::Log<hemelb::log::Info, hemelb::log::Singleton>("--> neighbouring data manager");
	neighbouringDataManager = new hemelb::geometry::neighbouring::NeighbouringDataManager(*latticeData,
				latticeData->GetNeighbouringData(),
				communicationNet);

	hemelb::log::Logger::Log<hemelb::log::Info, hemelb::log::Singleton>("--> lattice-Boltzmann model");
	latticeBoltzmannModel = new hemelb::lb::LBM<latticeType>(simConfig,
			&communicationNet,
			latticeData,
			simulationState,
			timings,
			neighbouringDataManager);

	hemelb::lb::MacroscopicPropertyCache& propertyCache = latticeBoltzmannModel->GetPropertyCache();

	hemelb::tau_ = latticeBoltzmannModel->GetLbmParams()->GetTau();

	if (simConfig->HasColloidSection()) {
		hemelb::log::Logger::Log<hemelb::log::Info, hemelb::log::Singleton>("--> colloid section present");

		timings[hemelb::reporting::Timers::colloidInitialisation].Start();
		hemelb::log::Logger::Log<hemelb::log::Info, hemelb::log::Singleton>("----> loading colloid configuration");
		std::string colloidConfigPath = simConfig->GetColloidConfigPath();
		hemelb::io::xml::Document xml(colloidConfigPath);

		hemelb::colloids::BodyForces::InitBodyForces(xml);
		hemelb::colloids::BoundaryConditions::InitBoundaryConditions(latticeData, xml);

		hemelb::log::Logger::Log<hemelb::log::Info, hemelb::log::Singleton>("----> initialising colloid controller");
		colloidController =
			new hemelb::colloids::ColloidController(*latticeData,
					*simulationState,
					simConfig,
					readGeometryData,
					xml,
					propertyCache,
					//latticeBoltzmannModel->GetLbmParams(),
					fileManager->GetColloidPath(),
					ioComms,
					timings);
		timings[hemelb::reporting::Timers::colloidInitialisation].Stop();
	}

	stabilityTester = new hemelb::lb::StabilityTester<latticeType>(latticeData,
			&communicationNet,
			simulationState,
			timings,
			monitoringConfig);
	entropyTester = NULL;

	if (monitoringConfig->doIncompressibilityCheck) {
		incompressibilityChecker = new hemelb::lb::IncompressibilityChecker <
		hemelb::net::PhasedBroadcastRegular<> > (latticeData,
				&communicationNet,
				simulationState,
				latticeBoltzmannModel->GetPropertyCache(),
				timings);
	} else {
		incompressibilityChecker = NULL;
	}

	inletValues = new hemelb::lb::iolets::BoundaryValues(hemelb::geometry::INLET_TYPE,
			latticeData,
			simConfig->GetInlets(),
			simulationState,
			ioComms,
			*unitConverter);

	outletValues = new hemelb::lb::iolets::BoundaryValues(hemelb::geometry::OUTLET_TYPE,
			latticeData,
			simConfig->GetOutlets(),
			simulationState,
			ioComms,
			*unitConverter);

	latticeBoltzmannModel->Initialise(inletValues, outletValues, unitConverter);

	//=======================================================================================
	// Check for GPU capabilities
	#ifdef HEMELB_USE_GPU
		check_GPU_capabilities();

		if(communicationNet.Rank()!=0) {
			bool res_InitGPU = latticeBoltzmannModel->Initialise_GPU(inletValues, outletValues, unitConverter);
			if (!res_InitGPU){
				printf("Rank: %d, Initialising the GPU failed... Abort... \n\n",communicationNet.Rank());
				Abort();	// Abort if initialiing the GPUs fail...
			}
		}
	#endif
	//=======================================================================================



	neighbouringDataManager->ShareNeeds();
	neighbouringDataManager->TransferNonFieldDependentInformation();

	propertyDataSource =
		new hemelb::extraction::LbDataSourceIterator(latticeBoltzmannModel->GetPropertyCache(),
				*latticeData,
				ioComms.Rank(),
				*unitConverter);

	if (simConfig->PropertyOutputCount() > 0) {

		for (unsigned outputNumber = 0; outputNumber < simConfig->PropertyOutputCount(); ++outputNumber) {
			simConfig->GetPropertyOutput(outputNumber)->filename = fileManager->GetDataExtractionPath()
					+ simConfig->GetPropertyOutput(outputNumber)->filename;
		}

		propertyExtractor = new hemelb::extraction::PropertyActor(*simulationState,
				simConfig->GetPropertyOutputs(),
				*propertyDataSource,
				timings, ioComms);
	}

	imagesPeriod = OutputPeriod(imagesPerSimulation);

	stepManager = new hemelb::net::phased::StepManager(2,
			&timings,
			hemelb::net::separate_communications);
	netConcern = new hemelb::net::phased::NetConcern(communicationNet);
	stepManager->RegisterIteratedActorSteps(*neighbouringDataManager, 0);

	if (colloidController != NULL) {
		stepManager->RegisterIteratedActorSteps(*colloidController, 1);
	}

	stepManager->RegisterIteratedActorSteps(*latticeBoltzmannModel, 1);

	stepManager->RegisterIteratedActorSteps(*inletValues, 1);
	stepManager->RegisterIteratedActorSteps(*outletValues, 1);
	stepManager->RegisterIteratedActorSteps(*stabilityTester, 1);

	if (entropyTester != NULL) {
		stepManager->RegisterIteratedActorSteps(*entropyTester, 1);
	}

	if (monitoringConfig->doIncompressibilityCheck) {
		stepManager->RegisterIteratedActorSteps(*incompressibilityChecker, 1);
	}

	if (propertyExtractor != NULL) {
		stepManager->RegisterIteratedActorSteps(*propertyExtractor, 1);
	}

	stepManager->RegisterCommsForAllPhases(*netConcern);

	hemelb::log::Logger::Log<hemelb::log::Info, hemelb::log::Singleton>("-------------------");
	hemelb::log::Logger::Log<hemelb::log::Info, hemelb::log::Singleton>("INITIALISE FINISHED");
}


// =============================================================================================
#ifdef HEMELB_USE_GPU
void SimulationMaster::check_GPU_capabilities()
{
	//hemelb::net::MpiCommunicator rank_Com;
	//proc_t myPiD = rank_Com.Rank(); // from units.h:  typedef int proc_t;

	int localRank  = communicationNet.Rank(); // Gives the local rank - change type to proc_t

	int dev_count=0;
	hipGetDeviceCount( &dev_count);
	// This function call returns 0 if there are no CUDA capable devices.
	if (dev_count == 0)
	{
		std::printf("--------------------------------------------------------------------------------\n");
		std::printf("Rank %d: There are no available device(s) that support CUDA... Need to Abort!!!\n", localRank);
		Abort();	//add an abort function here if no CUDA capable devices are detected
	}
	else {
		if(localRank==0) std::printf("Rank %d: Detected %d CUDA Capable device(s)\n", localRank, dev_count);
	}


	// Set the current GPU device
	hipError_t cudaStatus;
	int device;
	if(dev_count>1 && localRank!=0){
		cudaStatus = hipSetDevice((localRank-1)%dev_count);		//Set GPU - Rank 0 does not participate
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "GPU device setting failed\n");
			Abort();
			//return false;
		}
	}

	hipGetDevice(&device);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);
	// std::printf("Using device %d: %s - Assigned to Proc# %i \n\n", device, deviceProp.name, localRank);
	// if(localRank==0) printf("Using GPU device: %s \n\n", deviceProp.name);

	/*
	hipError_t cudaerr = hipDeviceSynchronize();
	if (cudaerr != hipSuccess)
	  printf("kernel launch failed with error \"%s\".\n",
		 hipGetErrorString(cudaerr));
	*/
}
#endif
// =============================================================================================


unsigned int SimulationMaster::OutputPeriod(unsigned int frequency) {
	if (frequency == 0) {
		return 1000000000;
	}

	unsigned long roundedPeriod = simulationState->GetTotalTimeSteps() / frequency;
	return hemelb::util::NumericalFunctions::max(1U, (unsigned int) roundedPeriod);
}

void SimulationMaster::HandleActors() {
	stepManager->CallActions();
}

void SimulationMaster::OnUnstableSimulation() {
	LogStabilityReport();
	hemelb::log::Logger::Log<hemelb::log::Warning, hemelb::log::Singleton>("ABORTING :: time step length: %f",
			simulationState->GetTimeStepLength());
	Finalise();
	Abort();
}

/**
 * Begin the simulation.
 */
void SimulationMaster::RunSimulation() {
	hemelb::log::Logger::Log<hemelb::log::Info, hemelb::log::Singleton>("SIMULATION STARTING");
	hemelb::log::Logger::Log<hemelb::log::Info, hemelb::log::Singleton>("-------------------");
	timings[hemelb::reporting::Timers::simulation].Start();

	while (simulationState->GetTimeStep() <= simulationState->GetTotalTimeSteps()) {
		DoTimeStep();

		if (simulationState->IsTerminating()) {
			break;
		}
	}

	timings[hemelb::reporting::Timers::simulation].Stop();
	Finalise();
}

void SimulationMaster::Finalise() {
	timings[hemelb::reporting::Timers::total].Stop();
	timings.Reduce();

#ifdef HEMELB_USE_GPU
	// Calls hipFree to delete the dynamically allocated memory on the GPU and hipStreamDestroy to delete the cuda streams
	// IOProc (RANK=0) does not allocate memory
	if (!IsCurrentProcTheIOProc()) {
		latticeBoltzmannModel->FinaliseGPU();
	}
#endif

	if (IsCurrentProcTheIOProc()) {
		reporter->FillDictionary();
		reporter->Write();
	}

	// DTMP: Logging output on communication as debug output for now.
	hemelb::log::Logger::Log<hemelb::log::Debug, hemelb::log::OnePerCore>("sync points: %lld, bytes sent: %lld",
			communicationNet.SyncPointsCounted,
			communicationNet.BytesSent);

	hemelb::log::Logger::Log<hemelb::log::Info, hemelb::log::Singleton>("-------------------");
	hemelb::log::Logger::Log<hemelb::log::Info, hemelb::log::Singleton>("SIMULATION FINISHED");
}


void SimulationMaster::DoTimeStep() {
	bool writeImage = ((simulationState->GetTimeStep() % imagesPeriod) == 0) ?
					true :
					false;

	if (simulationState->GetTimeStep() % 200 == 0) {
		hemelb::log::Logger::Log<hemelb::log::Info, hemelb::log::Singleton>("time step %07i :: write_image_to_disk %i",
				simulationState->GetTimeStep(),
				writeImage);
		LogStabilityReport();
	}

	RecalculatePropertyRequirements();

	HandleActors();


	// Check the stability of the code
	if (simulationState->GetStability() == hemelb::lb::Unstable) {
		OnUnstableSimulation();
	}

	// If the user requested to terminate converged steady flow simulations, mark
	// simulation to be finished at the end of the current timestep.
	if ((simulationState->GetStability() == hemelb::lb::StableAndConverged)
			&& monitoringConfig->convergenceTerminate) {
		LogStabilityReport();
		simulationState->SetIsTerminating(true);
	}

	// Colloid output
	//if ((simulationState->GetTimeStep() % 100 == 0) && colloidController != NULL)
	//	colloidController->OutputInformation(simulationState->GetTimeStep());

	if (simulationState->GetTimeStep() % FORCE_FLUSH_PERIOD == 0 && IsCurrentProcTheIOProc()) {
		fflush(NULL);
	}

	simulationState->Increment();
}

void SimulationMaster::RecalculatePropertyRequirements() {
	// Get the property cache & reset its list of properties to get.
	hemelb::lb::MacroscopicPropertyCache& propertyCache = latticeBoltzmannModel->GetPropertyCache();

	propertyCache.ResetRequirements();

	if (monitoringConfig->doIncompressibilityCheck) {
		propertyCache.densityCache.SetRefreshFlag();
		propertyCache.velocityCache.SetRefreshFlag();
	}

	// If extracting property results, check what's required by them.
	if (propertyExtractor != NULL) {
		propertyExtractor->SetRequiredProperties(propertyCache);
	}
}

/**
 * Called on error to abort the simulation and pull-down the MPI environment.
 */
void SimulationMaster::Abort() {
	// This gives us something to work from when we have an error - we get the rank
	// that calls abort, and we get a stack-trace from the exception having been thrown.
	hemelb::log::Logger::Log<hemelb::log::Critical, hemelb::log::Singleton>("ABORTING");
	hemelb::net::MpiEnvironment::Abort(1);

	exit(1);
}

void SimulationMaster::LogStabilityReport() {
	if (monitoringConfig->doIncompressibilityCheck
			&& incompressibilityChecker->AreDensitiesAvailable()) {
		hemelb::log::Logger::Log<hemelb::log::Info, hemelb::log::Singleton>("time step %07i :: tau: %.6f, max_relative_press_diff: %.3f, Ma: %.3f, max_vel_phys: %e",
				simulationState->GetTimeStep(),
				latticeBoltzmannModel->GetLbmParams()->GetTau(),
				incompressibilityChecker->GetMaxRelativeDensityDifference(),
				incompressibilityChecker->GetGlobalLargestVelocityMagnitude()
				/ hemelb::Cs,
				unitConverter->ConvertVelocityToPhysicalUnits(incompressibilityChecker->GetGlobalLargestVelocityMagnitude()));
	}

	if (simulationState->GetStability() == hemelb::lb::StableAndConverged) {
		hemelb::log::Logger::Log<hemelb::log::Info, hemelb::log::Singleton>("time step %07i :: steady flow simulation converged",
				simulationState->GetTimeStep());
	}
}
