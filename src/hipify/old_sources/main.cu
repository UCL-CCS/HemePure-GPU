#include "hip/hip_runtime.h"

// This file is part of HemeLB and is Copyright (C)
// the HemeLB team and/or their institutions, as detailed in the
// file AUTHORS. This software is provided under the terms of the
// license in the file LICENSE.

#include "net/mpi.h"
#include "net/IOCommunicator.h"
#include "configuration/CommandLine.h"
#include "SimulationMaster.h"

#include "net/MpiCommunicator.h"


int main(int argc, char *argv[])
{
  // main function needed to perform the entire simulation. Some
  // simulation parameters and performance statistics are output on
  // standard output

  // Bring up MPI
  hemelb::net::MpiEnvironment mpi(argc, argv);
  hemelb::log::Logger::Init();
  try
  {
    hemelb::net::MpiCommunicator commWorld = hemelb::net::MpiCommunicator::World();

    hemelb::net::IOCommunicator hemelbCommunicator(commWorld);
    try
    {
      // Parse command line
      hemelb::configuration::CommandLine options = hemelb::configuration::CommandLine(argc, argv);

      // Prepare main simulation object...
      SimulationMaster master = SimulationMaster(options, hemelbCommunicator);

	  // ..and run it.
      master.RunSimulation();

    }

    // Interpose this catch to print usage before propagating the error.
    catch (hemelb::configuration::CommandLine::OptionError& e)
    {
      hemelb::log::Logger::Log<hemelb::log::Critical, hemelb::log::Singleton>(hemelb::configuration::CommandLine::GetUsage());
      throw;
    }
  }
  catch (std::exception& e)
  {
    hemelb::log::Logger::Log<hemelb::log::Critical, hemelb::log::OnePerCore>(e.what());
    mpi.Abort(-1);
  }
  // MPI gets finalised by MpiEnv's d'tor.
}
